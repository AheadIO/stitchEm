// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm


#include <hip/hip_runtime.h>
#include <cstdio>

int main() {
  int devices;
  hipError_t err = hipGetDeviceCount(&devices);
  if (err != hipSuccess) {
    fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
    return 1;
  }

  if (devices == 0) {
    fprintf(stderr, "No CUDA device found!\n");
    return 1;
  }

  hipDeviceProp_t props_1;
  hipDeviceProp_t props_2;
  for (int i = 0; i < devices; ++i) {
    props_2 = props_1;
    err = hipGetDeviceProperties(&props_1, i);
    if (err != hipSuccess) {
      fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(err));
      return 1;
    }
    if (i > 0 && (props_1.major != props_2.major || props_1.minor != props_2.minor)) {
      fprintf(stderr, "Multiple CUDA arch not supported at the moment\n");
      return 1;
    }
  }

  printf("CUDA compute capability: %d%d", props_1.major, props_1.minor);

  // Easiest cross-platform way to pass the number seems to be the exit code
  // nvcc --run on windows seems to print the filename through cl.exe (that can't be silenced)
  return props_1.major * 10 + props_1.minor;
}
