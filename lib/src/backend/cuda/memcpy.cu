#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/memcpy.hpp"

#include "deviceBuffer.hpp"
#include "deviceStream.hpp"
#include "surface.hpp"

#include "backend/common/imageOps.hpp"

#include "cuda/util.hpp"

namespace VideoStitch {
namespace GPU {

__global__ void copyCubemapFace(uint32_t* __restrict__ src, int faceDim, int face, hipSurfaceObject_t surf) {
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < faceDim && y < faceDim) {
    uint32_t val = src[y * faceDim + x];
    uchar4 pix = make_uchar4(Image::RGBA::r(val), Image::RGBA::g(val), Image::RGBA::b(val), Image::RGBA::a(val));

    surfCubemapwrite(pix, surf, (int)x * sizeof(uchar4), (int)y, face);
  }
}

Status memcpyCubemapAsync(CubemapSurface& cubemapSurface, Buffer<uint32_t> xPosPbo, Buffer<uint32_t> xNegPbo,
                          Buffer<uint32_t> yPosPbo, Buffer<uint32_t> yNegPbo, Buffer<uint32_t> zPosPbo,
                          Buffer<uint32_t> zNegPbo, size_t faceDim, const Stream& stream) {
  dim3 block(16, 16, 1);
  dim3 grid((unsigned)Cuda::ceilDiv(faceDim, block.x), (unsigned)Cuda::ceilDiv(faceDim, block.y), 1);

  copyCubemapFace<<<grid, block, 0, stream.get()>>>(xPosPbo.get(), (int)faceDim, 0, cubemapSurface.get().surface());
  copyCubemapFace<<<grid, block, 0, stream.get()>>>(xNegPbo.get(), (int)faceDim, 1, cubemapSurface.get().surface());
  copyCubemapFace<<<grid, block, 0, stream.get()>>>(yPosPbo.get(), (int)faceDim, 2, cubemapSurface.get().surface());
  copyCubemapFace<<<grid, block, 0, stream.get()>>>(yNegPbo.get(), (int)faceDim, 3, cubemapSurface.get().surface());
  copyCubemapFace<<<grid, block, 0, stream.get()>>>(zPosPbo.get(), (int)faceDim, 4, cubemapSurface.get().surface());
  copyCubemapFace<<<grid, block, 0, stream.get()>>>(zNegPbo.get(), (int)faceDim, 5, cubemapSurface.get().surface());

  return CUDA_STATUS;
}

__global__ void resetArrayKernel(hipSurfaceObject_t dst, size_t width, size_t height) {
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    surf2Dwrite(0, dst, x * sizeof(uint32_t), y);
  }
}

Status memsetToZeroAsync(Surface& dst, const Stream& stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(dst.width(), 16), (unsigned)Cuda::ceilDiv(dst.height(), 16), 1);

  resetArrayKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), dst.width(), dst.height());
  return CUDA_STATUS;
}

}  // namespace GPU
}  // namespace VideoStitch
