// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/core1/mergerKernel.hpp"

#include "../deviceBuffer.hpp"
#include "../deviceStream.hpp"

#include "backend/common/imageOps.hpp"
#include "cuda/error.hpp"
#include "cuda/util.hpp"
#include "libvideostitch/panoDef.hpp"
#include "core1/imageMapping.hpp"

namespace VideoStitch {

namespace {

// CUDA/OpenCL shared implementation
#include "../gpuKernelDef.h"

#include <backend/common/core1/mergerKernel.gpu>
}  // namespace

namespace Core {

Status countInputs(TextureTarget t, const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo, const ImageMapping& fromIm,
                   GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  if (fromIm.getOutputRect(t).empty()) {
    return Status::OK();
  }

  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getHeight(), dimBlock.y), 1);
  if (fromIm.getOutputRect(t).right() >= (int64_t)pano.getWidth()) {
    countInputsKernelWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  } else {
    countInputsKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  }
  return CUDA_STATUS;
}

Status colorMap(const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  const int64_t size = pano.getWidth() * pano.getHeight();
  dim3 dimBlock(512);
  dim3 dimGrid(Cuda::compute2DGridForFlatBuffer(size, 512));
  colormapKernel<<<dimGrid, dimBlock, 0, stream>>>(pbo.get(), (unsigned)pano.getWidth(), (unsigned)size);
  return CUDA_ERROR(hipStreamSynchronize(stream));
}

Status stitchingError(TextureTarget t, const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo,
                      const ImageMapping& fromIm, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  if (fromIm.getOutputRect(t).empty()) {
    return Status::OK();
  }

  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getHeight(), dimBlock.y), 1);
  if (fromIm.getOutputRect(t).right() >= (int64_t)pano.getWidth()) {
    stitchingErrorKernelWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  } else {
    stitchingErrorKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  }
  return CUDA_STATUS;
}

Status exposureDiffRGB(TextureTarget t, const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo,
                       const ImageMapping& fromIm, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  if (fromIm.getOutputRect(t).empty()) {
    return Status::OK();
  }

  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getHeight(), dimBlock.y), 1);
  if (fromIm.getOutputRect(t).right() >= (int64_t)pano.getWidth()) {
    exposureErrorRGBKernelWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  } else {
    exposureErrorRGBKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  }
  return CUDA_STATUS;
}

Status amplitude(const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  const int64_t size = pano.getWidth() * pano.getHeight();
  dim3 dimBlock(512);
  dim3 dimGrid(Cuda::compute2DGridForFlatBuffer(size, 512));
  amplitudeKernel<<<dimGrid, dimBlock, 0, stream>>>(pbo.get(), 0, (3 * 256 * 256), (unsigned)pano.getWidth(),
                                                    (unsigned)size);
  return CUDA_ERROR(hipStreamSynchronize(stream));
}

Status disregardNoDiffArea(const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  const int64_t size = pano.getWidth() * pano.getHeight();
  dim3 dimBlock(512);
  dim3 dimGrid(Cuda::compute2DGridForFlatBuffer(size, 512));
  maskOutSingleInput<<<dimGrid, dimBlock, 0, stream>>>(pbo.get(), (unsigned)pano.getWidth(), (unsigned)size);
  return CUDA_ERROR(hipStreamSynchronize(stream));
}

Status checkerMerge(TextureTarget t, const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo, const ImageMapping& fromIm,
                    unsigned checkerSize, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  if (fromIm.getOutputRect(t).empty()) {
    return Status::OK();
  }
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getHeight(), dimBlock.y), 1);
  if (fromIm.getOutputRect(t).right() >= (int64_t)pano.getWidth()) {
    checkerInsertKernelWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top(), checkerSize);
  } else {
    checkerInsertKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top(), checkerSize);
  }
  return CUDA_STATUS;
}

Status noblend(TextureTarget t, const PanoDefinition& pano, GPU::Buffer<uint32_t> pbo, const ImageMapping& fromIm,
               GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  if (fromIm.getOutputRect(t).empty()) {
    return Status::OK();
  }

  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(fromIm.getOutputRect(t).getHeight(), dimBlock.y), 1);
  if (fromIm.getOutputRect(t).right() >= (int64_t)pano.getWidth()) {
    noblendKernelWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  } else {
    noblendKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(
        pbo.get(), (unsigned)pano.getWidth(), (unsigned)pano.getHeight(), fromIm.getDeviceOutputBuffer(t).get(),
        (unsigned)fromIm.getOutputRect(t).getWidth(), (unsigned)fromIm.getOutputRect(t).getHeight(),
        (unsigned)fromIm.getOutputRect(t).left(), (unsigned)fromIm.getOutputRect(t).top());
  }
  return CUDA_STATUS;
}
}  // namespace Core
}  // namespace VideoStitch
