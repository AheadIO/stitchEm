#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/core1/strip.hpp"

#include "../deviceBuffer.hpp"
#include "../deviceStream.hpp"
#include "../core/transformStack.cu"

#include "cuda/util.hpp"
#include "core/geoTransform.hpp"
#include "core/transformGeoParams.hpp"

#include <limits>

namespace VideoStitch {
namespace Core {
enum Direction { Vertical, Horizontal };

template <Convert2D3DFnT toSphere, Direction direction>
__global__ void stripKernel(unsigned char* dstBuf, int2 dstDim, float2 distCenter, const vsDistortion distortion,
                            const float2 inputScale, float min, float max) {
  int2 dst = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

  if (dst.x < dstDim.x && dst.y < dstDim.y) {
    /* extract only if in the strip */
    float2 sph;
    sph.x = (float)dst.x - dstDim.x / 2;
    sph.y = (float)dst.y - dstDim.y / 2;
    sph.x -= distCenter.x;
    sph.y -= distCenter.y;

    // inverseRadial(sph, radial0, radial1, radial2, radial3, radial4);
    sph.x /= inputScale.x;
    sph.y /= inputScale.y;

    /* take input projection into account*/
    float3 pt = toSphere(sph);

    if (direction == Horizontal) {
      vsfloat3x3 rot;
      rot.values[0][0] = 0.0f;
      rot.values[0][1] = -1.0f;
      rot.values[0][2] = 0.0f;
      rot.values[1][0] = 1.0f;
      rot.values[1][1] = 0.0f;
      rot.values[1][2] = 0.0f;
      rot.values[2][0] = 0.0f;
      rot.values[2][1] = 0.0f;
      rot.values[2][2] = 1.0f;
      pt = rotateSphere(pt, rot);
    }

    sph = SphereToErect(pt);
    if (sph.x < min || sph.x > max) {
      dstBuf[dstDim.x * dst.y + dst.x] = 1;
    }
  }
}

#define STRIP_KERNEL(transformFn, direction)                             \
  stripKernel<transformFn, direction><<<dimGrid, dimBlock, 0, stream>>>( \
      dst.get(), dstDim, distCenter, geoParams.getDistortion(), inputScale, min, max);

Status hStrip(GPU::Buffer<unsigned char> dst, std::size_t dstWidth, std::size_t dstHeight, float min, float max,
              InputDefinition::Format fmt, float distCenterX, float distCenterY, const TransformGeoParams& geoParams,
              const float2& inputScale, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
  int2 dstDim = make_int2((int)dstWidth, (int)dstHeight);

  // hardcode some stuff while we don't know how to evaluate the rig
  float2 distCenter = make_float2(distCenterX, distCenterY);
  switch (fmt) {
    case InputDefinition::Format::Rectilinear:
      STRIP_KERNEL(RectToSphere, Horizontal);
      break;
    case InputDefinition::Format::Equirectangular:
      STRIP_KERNEL(ErectToSphere, Horizontal);
      break;
    case InputDefinition::Format::CircularFisheye:
    case InputDefinition::Format::FullFrameFisheye:
    case InputDefinition::Format::CircularFisheye_Opt:
    case InputDefinition::Format::FullFrameFisheye_Opt:
      STRIP_KERNEL(FisheyeToSphere, Horizontal);
      break;
  }
  return CUDA_STATUS;
}

Status vStrip(GPU::Buffer<unsigned char> dst, std::size_t dstWidth, std::size_t dstHeight, float min, float max,
              InputDefinition::Format fmt, float distCenterX, float distCenterY, const TransformGeoParams& geoParams,
              const float2& inputScale, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
  int2 dstDim = make_int2((int)dstWidth, (int)dstHeight);

  // hardcode some stuff while we don't know how to evaluate the rig
  float2 distCenter = make_float2(distCenterX, distCenterY);
  switch (fmt) {
    case InputDefinition::Format::Rectilinear:
      STRIP_KERNEL(RectToSphere, Vertical);
      break;
    case InputDefinition::Format::Equirectangular:
      STRIP_KERNEL(ErectToSphere, Vertical);
      break;
    case InputDefinition::Format::CircularFisheye:
    case InputDefinition::Format::FullFrameFisheye:
    case InputDefinition::Format::CircularFisheye_Opt:
    case InputDefinition::Format::FullFrameFisheye_Opt:
      STRIP_KERNEL(FisheyeToSphere, Vertical);
      break;
  }
  return CUDA_STATUS;
}
}  // namespace Core
}  // namespace VideoStitch
