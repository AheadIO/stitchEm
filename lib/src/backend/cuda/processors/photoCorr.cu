#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/processors/photoCorr.hpp"

#include "../surface.hpp"
#include "../deviceStream.hpp"

#include "backend/common/imageOps.hpp"
#include "cuda/util.hpp"
#include "core/kernels/photoStack.cu"

namespace VideoStitch {
namespace Core {

template <class PhotoCorrection>
__global__ void preStitchPhotoCorrectionKernel(hipSurfaceObject_t buffer, const int width, const int height,
                                               const float rMult, const float gMult, const float bMult,
                                               const float vigCenterX, const float vigCenterY,
                                               const float inverseDemiDiagonalSquared, const float vigCoeff0,
                                               const float vigCoeff1, const float vigCoeff2, const float vigCoeff3,
                                               const TransformPhotoParam photoParam) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const float fx = (float)x - (float)width / 2.0f + 0.5f;  // FIXME: do we need to apply center shift here ?
    const float fy = (float)y - (float)height / 2.0f + 0.5f;
    /**
     * Compute vignetting:
     * Done before uv is shifted for texture fetch.
     * vigMult = a0 + a1 * r + a2 * r^2 + a3 * r^3
     *         = a0 + r * (a1 + r * (a2 + r * a3))
     */
    const float vigRadiusSquared =
        ((fx - vigCenterX) * (fx - vigCenterX) + (fy - vigCenterY) * (fy - vigCenterY)) * inverseDemiDiagonalSquared;
    float vigMult = vigRadiusSquared * vigCoeff3;
    vigMult += vigCoeff2;
    vigMult *= vigRadiusSquared;
    vigMult += vigCoeff1;
    vigMult *= vigRadiusSquared;
    vigMult += vigCoeff0;
    vigMult = 1.0f / vigMult;

    uint32_t v;
    surf2Dread(&v, buffer, x * sizeof(uint32_t), y);
    float3 color = make_float3(Image::RGBA::r(v), Image::RGBA::g(v), Image::RGBA::b(v));

    // exposure correction
    color = PhotoCorrection::corr(color, photoParam.floatParam, (float*)photoParam.transformData);
    color.x *= rMult * vigMult;
    color.y *= gMult * vigMult;
    color.z *= bMult * vigMult;
    color = PhotoCorrection::invCorr(color, photoParam.floatParam, (float*)photoParam.transformData);

    uint32_t pixel = Image::RGBA::pack(Image::clamp8(__float2int_rn(color.x)), Image::clamp8(__float2int_rn(color.y)),
                                       Image::clamp8(__float2int_rn(color.z)), Image::RGBA::a(v));
    surf2Dwrite(pixel, buffer, x * sizeof(uint32_t), y);
  }
}

#define RUN_KERNEL(PhotoCorrection)                                                                                   \
  preStitchPhotoCorrectionKernel<PhotoCorrection><<<dimGrid, dimBlock, 0, stream.get()>>>(                            \
      buffer.get().surface(), width, height, rMult, gMult, bMult, vigCenterX, vigCenterY, inverseDemiDiagonalSquared, \
      vigCoeff0, vigCoeff1, vigCoeff2, vigCoeff3, photoParam);

Status linearPhotoCorrection(GPU::Surface& buffer, const int width, const int height, const float rMult,
                             const float gMult, const float bMult, const float vigCenterX, const float vigCenterY,
                             const float inverseDemiDiagonalSquared, const float vigCoeff0, const float vigCoeff1,
                             const float vigCoeff2, const float vigCoeff3, const TransformPhotoParam& photoParam,
                             GPU::Stream stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  RUN_KERNEL(LinearPhotoCorrection);
  return CUDA_STATUS;
}

Status gammaPhotoCorrection(GPU::Surface& buffer, const int width, const int height, const float rMult,
                            const float gMult, const float bMult, const float vigCenterX, const float vigCenterY,
                            const float inverseDemiDiagonalSquared, const float vigCoeff0, const float vigCoeff1,
                            const float vigCoeff2, const float vigCoeff3, const TransformPhotoParam& photoParam,
                            GPU::Stream stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  RUN_KERNEL(GammaPhotoCorrection);
  return CUDA_STATUS;
}

Status emorPhotoCorrection(GPU::Surface& buffer, const int width, const int height, const float rMult,
                           const float gMult, const float bMult, const float vigCenterX, const float vigCenterY,
                           const float inverseDemiDiagonalSquared, const float vigCoeff0, const float vigCoeff1,
                           const float vigCoeff2, const float vigCoeff3, const TransformPhotoParam& photoParam,
                           GPU::Stream stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  RUN_KERNEL(EmorPhotoCorrection);
  return CUDA_STATUS;
}
#undef RUN_KERNEL
}  // namespace Core
}  // namespace VideoStitch
