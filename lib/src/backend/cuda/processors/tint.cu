#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/processors/tint.hpp"

#include "backend/common/imageOps.hpp"
#include "backend/cuda/surface.hpp"
#include "backend/cuda/deviceStream.hpp"
#include "cuda/util.hpp"

namespace VideoStitch {
namespace Core {

namespace {
/**
 * A kernel that tints everything with a color.
 */
__global__ void tintKernel(hipSurfaceObject_t dst, unsigned width, unsigned height, int32_t r, int32_t g, int32_t b) {
  const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    uint32_t srcColor;
    surf2Dread(&srcColor, dst, x * sizeof(uint32_t), y);
    // 0.2126 R + 0.7152 G + 0.0722 B
    const int32_t luminosity =
        1742 * Image::RGBA::r(srcColor) + 5859 * Image::RGBA::g(srcColor) + 591 * Image::RGBA::b(srcColor);
    uint32_t dstColor = Image::RGBA::pack((r * luminosity) >> 21, (g * luminosity) >> 21, (b * luminosity) >> 21,
                                          Image::RGBA::a(srcColor));
    surf2Dwrite(dstColor, dst, x * sizeof(uint32_t), y);
  }
}
}  // namespace

Status tint(GPU::Surface& dst, unsigned width, unsigned height, int32_t r, int32_t g, int32_t b, GPU::Stream stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  tintKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), width, height, r, g, b);
  return CUDA_STATUS;
}
}  // namespace Core
}  // namespace VideoStitch
