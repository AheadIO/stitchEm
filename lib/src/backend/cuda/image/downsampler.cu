// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/image/downsampler.hpp"
#include "gpu/2dBuffer.hpp"

#include "../deviceBuffer2D.hpp"
#include "../deviceStream.hpp"
#include "../surface.hpp"
#include "../gpuKernelDef.h"

#include "cuda/util.hpp"

#include "backend/cuda/core1/kernels/defKernel.cu"

#include "backend/common/image/downsampler.gpu"

#include <hip/hip_runtime.h>
#include <cassert>

namespace VideoStitch {
namespace Image {

Status downsample(GPU::Buffer2D& in, GPU::Buffer2D& out, GPU::Stream stream) {
  assert(in.getWidth() / out.getWidth() == in.getHeight() / out.getHeight());
  int factor = (int)(in.getWidth() / out.getWidth());
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(out.getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(out.getHeight(), dimBlock.y), 1);
  downsamplePlanarKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      out.get().raw(), (unsigned)out.getPitch(), in.get().raw(), (unsigned)in.getPitch(), (unsigned)in.getWidth(),
      (unsigned)in.getHeight(), factor);
  return CUDA_STATUS;
}

Status downsampleRGBASurf2x(GPU::Surface& dst, const GPU::Surface& src, unsigned dstWidth, unsigned dstHeight,
                            GPU::Stream stream) {
  if (src.width() != 2 * dst.width() || src.height() != 2 * dst.height()) {
    return {Origin::GPU, ErrType::UnsupportedAction, "Downsampling RGBA surfaces only implemented for even dimensions"};
  }
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
  downsampleRGBASurfKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), src.get().texture(), dstWidth,
                                                                   dstHeight);
  return CUDA_STATUS;
}

Status downsampleRGBA(int factor, GPU::Buffer2D& in, GPU::Buffer2D& out, GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(out.getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(out.getHeight(), dimBlock.y), 1);
  downsampleRGBAKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      out.get().raw(), (unsigned)out.getPitch(), in.get().raw(), (unsigned)in.getPitch(), (unsigned)in.getWidth(),
      (unsigned)in.getHeight(), factor);
  return CUDA_STATUS;
}

Status downsampleRGB(int factor, GPU::Buffer2D& in, GPU::Buffer2D& out, GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(out.getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(out.getHeight(), dimBlock.y), 1);
  downsampleRGBKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(out.get(), (unsigned)out.getPitch(), in.get().raw(),
                                                              (unsigned)in.getPitch(), (unsigned)in.getWidth(),
                                                              (unsigned)in.getHeight(), factor);
  return CUDA_STATUS;
}

Status downsampleYUV422(int factor, GPU::Buffer2D& in, GPU::Buffer2D& out, GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  // each thread accumulates for 2 (horizontally) consecutive destination pixels
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(out.getWidth() / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv(out.getHeight(), dimBlock.y), 1);
  downsampleYUV422Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      out.get().raw(), (unsigned)out.getPitch(), in.get().raw(), (unsigned)in.getPitch(), (unsigned)in.getWidth(),
      (unsigned)in.getHeight(), factor);
  return CUDA_STATUS;
}

Status downsampleYUV422P10(int factor, GPU::Buffer2D& yIn, GPU::Buffer2D& uIn, GPU::Buffer2D& vIn, GPU::Buffer2D& yOut,
                           GPU::Buffer2D& uOut, GPU::Buffer2D& vOut, GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(yOut.getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(yOut.getHeight(), dimBlock.y), 1);
  downsampleYUV422P10Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      yOut.get().raw(), (unsigned)yOut.getPitch(), uOut.get().raw(), (unsigned)uOut.getPitch(), vOut.get().raw(),
      (unsigned)vOut.getPitch(), yIn.get().raw(), (unsigned)yIn.getPitch(), uIn.get().raw(), (unsigned)uIn.getPitch(),
      vIn.get().raw(), (unsigned)vIn.getPitch(), (unsigned)yIn.getWidth(), (unsigned)yIn.getHeight(), factor);
  return CUDA_STATUS;
}

Status downsampleYV12(int factor, GPU::Buffer2D& yIn, GPU::Buffer2D& uIn, GPU::Buffer2D& vIn, GPU::Buffer2D& yOut,
                      GPU::Buffer2D& uOut, GPU::Buffer2D& vOut, GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(yOut.getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(yOut.getHeight(), dimBlock.y), 1);
  downsampleYV12Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      yOut.get().raw(), (unsigned)yOut.getPitch(), uOut.get().raw(), (unsigned)uOut.getPitch(), vOut.get().raw(),
      (unsigned)vOut.getPitch(), yIn.get().raw(), (unsigned)yIn.getPitch(), uIn.get().raw(), (unsigned)uIn.getPitch(),
      vIn.get().raw(), (unsigned)vIn.getPitch(), (unsigned)yIn.getWidth(), (unsigned)yIn.getHeight(), factor);
  return CUDA_STATUS;
}

Status downsampleNV12(int factor, GPU::Buffer2D& yIn, GPU::Buffer2D& uvIn, GPU::Buffer2D& yOut, GPU::Buffer2D& uvOut,
                      GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(yOut.getWidth(), dimBlock.x),
                     (unsigned)Cuda::ceilDiv(yOut.getHeight(), dimBlock.y), 1);
  downsampleNV12Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      yOut.get().raw(), (unsigned)yOut.getPitch(), uvOut.get().raw(), (unsigned)uvOut.getPitch(), yIn.get().raw(),
      (unsigned)yIn.getPitch(), uvIn.get().raw(), (unsigned)uvIn.getPitch(), (unsigned)yIn.getWidth(),
      (unsigned)yIn.getHeight(), factor);
  return CUDA_STATUS;
}
}  // namespace Image
}  // namespace VideoStitch
