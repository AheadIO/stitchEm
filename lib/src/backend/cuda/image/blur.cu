#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/image/blur.hpp"

#include "../deviceBuffer.hpp"
#include "../deviceStream.hpp"

#include "cuda/util.hpp"
#include "image/transpose.hpp"

#include "libvideostitch/profile.hpp"

#include <hip/hip_runtime.h>
#include <cassert>

#define RGBA_BOX_BLUR_1D_BLOCK_SIZE (4 * 32)
#define RGBA_BOX_BLUR_SS_1D_BLOCK_SIZE (4 * 32)

template <typename Type>
struct ScalarPixel {
  typedef Type T;
};

#include "image/kernels/blurKernel.cu"
#include "image/kernels/blurKernelSmallSupport.cu"
#include "image/kernels/unrolledGaussianKernels.cu"

namespace VideoStitch {
namespace Image {

namespace {
template <typename T>
void swap(T& a, T& b) {
  T tmp = a;
  a = b;
  b = tmp;
}
}  // namespace

template <typename T>
Status boxBlur1DNoWrap(GPU::Buffer<T> dst, GPU::Buffer<const T> src, std::size_t width, std::size_t height,
                       unsigned radius, unsigned blockSize, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(blockSize, 1, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), 1, 1);
  if ((std::size_t)radius >= height) {
    blur1DKernelNoWrapHugeRadius<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                                   (unsigned)height, radius);
  } else if ((std::size_t)(2 * radius) >= height) {
    blur1DKernelNoWrapLargeRadius<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                                    (unsigned)height, radius);
  } else if (COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= radius) {  // if radius is short enough for blurColumnsKernel
    dim3 blocks((unsigned)Cuda::ceilDiv(width, COLUMNS_BLOCKDIM_X),
                (unsigned)Cuda::ceilDiv(height, (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y)));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
    blurColumnsKernelNoWrap<T><<<blocks, threads, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                               (unsigned)height, (unsigned)width, radius);
  } else {
    blur1DKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                         (unsigned)height, radius);
  }
  return CUDA_STATUS;
}

template <typename T>
Status boxBlur1DWrap(GPU::Buffer<T> dst, GPU::Buffer<const T> src, std::size_t width, std::size_t height,
                     unsigned radius, unsigned blockSize, GPU::Stream stream) {
  if ((std::size_t)(2 * radius) >= height) {
    // the blur takes the whole buffer for all pixels since the stencil is larger than the patchlet,
    // so just resize the stencil
    radius = (unsigned)(height / 2 - 1);
  }

  if (COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= radius) {  // if radius is short enough for blurColumnsKernel
    dim3 blocks((unsigned)Cuda::ceilDiv(width, COLUMNS_BLOCKDIM_X),
                (unsigned)Cuda::ceilDiv(height, (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y)));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
    blurColumnsKernelWrap<<<blocks, threads, 0, stream.get()>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                                (unsigned)height, (unsigned)width, radius);
  } else {
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), 1, 1);
    blur1DKernelWrap<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                             (unsigned)height, radius);
  }
  return CUDA_STATUS;
}

template Status boxBlur1DNoWrap(GPU::Buffer<float> dst, GPU::Buffer<const float> src, std::size_t width,
                                std::size_t height, unsigned radius, unsigned blockSize, GPU::Stream stream);

template Status boxBlur1DNoWrap(GPU::Buffer<float2> dst, GPU::Buffer<const float2> src, std::size_t width,
                                std::size_t height, unsigned radius, unsigned blockSize, GPU::Stream stream);
template Status boxBlur1DNoWrap(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                                std::size_t height, unsigned radius, unsigned blockSize, GPU::Stream stream);

template Status boxBlur1DWrap(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, unsigned radius, unsigned blockSize, GPU::Stream stream);

template <typename T>
Status gaussianBlur2D(GPU::Buffer<T> dst, GPU::Buffer<const T> src, GPU::Buffer<T> work, std::size_t width,
                      std::size_t height, unsigned radius, unsigned passes, bool wrap, GPU::Stream stream) {
  assert(passes > 0);
  const unsigned blockSize = RGBA_BOX_BLUR_1D_BLOCK_SIZE;
  // First pass is from src to work;
  PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(work, src, width, height, radius, blockSize, stream));
  // Other passes ping-pong between work buffers.
  GPU::Buffer<T> srcBuf = work;
  GPU::Buffer<T> dstBuf = dst;
  for (unsigned i = 1; i < passes; ++i) {
    PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(dstBuf, srcBuf.as_const(), width, height, radius, blockSize, stream));
    swap(dstBuf, srcBuf);
  }
  // transpose
  PROPAGATE_FAILURE_STATUS(transpose(dstBuf.get().raw(), srcBuf.get().raw(), width, height, stream));
  swap(dstBuf, srcBuf);
  if (wrap) {
    for (unsigned i = 0; i < passes; ++i) {
      PROPAGATE_FAILURE_STATUS(boxBlur1DWrap(dstBuf, srcBuf.as_const(), height, width, radius, blockSize, stream));
      swap(dstBuf, srcBuf);
    }
  } else {
    for (unsigned i = 0; i < passes; ++i) {
      PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(dstBuf, srcBuf.as_const(), height, width, radius, blockSize, stream));
      swap(dstBuf, srcBuf);
    }
  }
  PROPAGATE_FAILURE_STATUS(transpose(dstBuf.get().raw(), srcBuf.get().raw(), height, width, stream));
  // There are (passes - 1) swaps, then the transpose swap, then passes swaps.
  // i.e. 2 * passes swaps. So overall srcBuf ad dstBuff are unchanged from their first state.
  assert(dstBuf == dst);
  return CUDA_STATUS;
}

template Status gaussianBlur2D(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src,
                               GPU::Buffer<unsigned char> work, std::size_t width, std::size_t height, unsigned radius,
                               unsigned passes, bool wrap, GPU::Stream stream);
template Status gaussianBlur2D(GPU::Buffer<float2> dst, GPU::Buffer<const float2> src, GPU::Buffer<float2> work,
                               std::size_t width, std::size_t height, unsigned radius, unsigned passes, bool wrap,
                               GPU::Stream stream);

Status boxBlurColumnsWrapRGBA210(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t width,
                                 std::size_t height, unsigned radius, GPU::Stream stream) {
  if ((std::size_t)(2 * radius) >= height) {
    // the blur takes the whole buffer for all pixels since the stencil is larger than the patchlet,
    // so just resize the stencil
    radius = (unsigned)(height / 2 - 1);
  }
  if (COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= radius) {  // if radius is short enough for blurColumnsKernel
    dim3 blocks((unsigned)Cuda::ceilDiv(width, COLUMNS_BLOCKDIM_X),
                (unsigned)Cuda::ceilDiv(height, (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y)));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
    blurColumnsKernelWrap<uint32_t><<<blocks, threads, 0, stream.get()>>>(
        dst.get().raw(), src.get().raw(), (unsigned)width, (unsigned)height, (unsigned)width, radius);
  } else {
    dim3 dimBlock(RGBA_BOX_BLUR_1D_BLOCK_SIZE, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), 1, 1);
    blur1DKernelWrap<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                             (unsigned)height, radius);
  }
  return CUDA_STATUS;
}

Status boxBlurColumnsNoWrapRGBA210(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t width,
                                   std::size_t height, unsigned radius, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(RGBA_BOX_BLUR_1D_BLOCK_SIZE, 1, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), 1, 1);
  if ((std::size_t)radius >= height) {
    blur1DKernelNoWrapHugeRadius<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                                   (unsigned)height, radius);
  } else if ((std::size_t)(2 * radius) >= height) {
    blur1DKernelNoWrapLargeRadius<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                                    (unsigned)height, radius);
  } else if (COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= radius) {  // if radius is short enough for blurColumnsKernel
    dim3 blocks((unsigned)Cuda::ceilDiv(width, COLUMNS_BLOCKDIM_X),
                (unsigned)Cuda::ceilDiv(height, (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y)));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
    blurColumnsKernelNoWrap<uint32_t><<<blocks, threads, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                                      (unsigned)height, (unsigned)width, radius);
  } else {
    blur1DKernelNoWrap<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)width,
                                                         (unsigned)height, radius);
  }
  return CUDA_STATUS;
}

Status boxBlurRowsRGBA210(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t width,
                          std::size_t height, unsigned radius, GPU::Stream stream, bool wrap) {
  dim3 blocks((unsigned)Cuda::ceilDiv(width, (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X)),
              (unsigned)Cuda::ceilDiv(height, ROWS_BLOCKDIM_Y));
  dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
  if ((std::size_t)(2 * radius) >= width) {
    // the blur takes the whole buffer for all pixels since the stencil is larger than the patchlet,
    // so just resize the stencil
    radius = (unsigned)(width / 2 - 1);
  }
  if (wrap) {
    blurRowsKernelWrap<<<blocks, threads, 0, stream.get()>>>(dst.get().raw(), src.get().raw(), width, height, width,
                                                             radius);
  } else {
    blurRowsKernelNoWrap<<<blocks, threads, 0, stream.get()>>>(dst.get().raw(), src.get().raw(), width, height, width,
                                                               radius);
  }
  return CUDA_STATUS;
}

Status gaussianBlur2DRGBA210(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, GPU::Buffer<uint32_t> work,
                             std::size_t width, std::size_t height, unsigned radius, unsigned passes, bool wrap,
                             GPU::Stream stream) {
  assert(passes > 0);
  // First pass is from src to work;
  PROPAGATE_FAILURE_STATUS(boxBlurColumnsNoWrapRGBA210(work, src, width, height, radius, stream));
  // Other passes ping-pong between work buffers.
  GPU::Buffer<uint32_t> srcBuf = work;
  GPU::Buffer<uint32_t> dstBuf = dst;
  for (unsigned i = 1; i < passes; ++i) {
    PROPAGATE_FAILURE_STATUS(boxBlurColumnsNoWrapRGBA210(dstBuf, srcBuf.as_const(), width, height, radius, stream));
    swap(dstBuf, srcBuf);
  }
  if ((ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= radius) &&
      ((std::size_t)2 * radius < height)) {  // boxBlurRowsRGBA210 works only in this case
    for (unsigned i = 0; i < passes; ++i) {
      PROPAGATE_FAILURE_STATUS(boxBlurRowsRGBA210(dstBuf, srcBuf.as_const(), width, height, radius, stream, wrap));
      swap(dstBuf, srcBuf);
    }
    swap(dstBuf, srcBuf);
    assert(dstBuf == dst);
  } else {
    // transpose
    PROPAGATE_FAILURE_STATUS(transpose(dstBuf.get().raw(), srcBuf.get().raw(), width, height, stream));
    swap(dstBuf, srcBuf);
    if (wrap) {
      for (unsigned i = 0; i < passes; ++i) {
        PROPAGATE_FAILURE_STATUS(boxBlurColumnsWrapRGBA210(dstBuf, srcBuf.as_const(), height, width, radius, stream));
        swap(dstBuf, srcBuf);
      }
    } else {
      for (unsigned i = 0; i < passes; ++i) {
        PROPAGATE_FAILURE_STATUS(boxBlurColumnsNoWrapRGBA210(dstBuf, srcBuf.as_const(), height, width, radius, stream));
        swap(dstBuf, srcBuf);
      }
    }
    PROPAGATE_FAILURE_STATUS(transpose(dstBuf.get().raw(), srcBuf.get().raw(), height, width, stream));
    // There are (passes - 1) swaps, then the transpose swap, then passes swaps.
    // i.e. 2 * passes swaps. So overall srcBuf ad dstBuff are unchanged from their first state.
    assert(dstBuf == dst);
  }
  return CUDA_STATUS;
}

Status gaussianBlur1DRGBA210SS(uint32_t* dst, const uint32_t* src, std::size_t width, std::size_t height,
                               unsigned radius, bool wrap, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  // Block organization is as follows for a 5x3 image and dimBlock.x == 3
  //  00 00 00 10 10
  //  01 01 01 11 11
  //  02 02 02 12 12
  // Handle the interior
  if ((unsigned)width > 2 * radius) {
    dim3 dimBlock(RGBA_BOX_BLUR_SS_1D_BLOCK_SIZE, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(width - 2 * radius, dimBlock.x), (unsigned)height, 1);
    assert(2 * radius < dimBlock.x);
    switch (radius) {
      case 1:
        gaussianBlur1DRGBA210SSKernelInterior<unrolledGaussianKernel1>
            <<<dimGrid, dimBlock, 16 * (dimBlock.x + 2 * radius), stream>>>(dst, src, (unsigned)width, (unsigned)height,
                                                                            radius);
        break;
      case 2:
        gaussianBlur1DRGBA210SSKernelInterior<unrolledGaussianKernel2>
            <<<dimGrid, dimBlock, 16 * (dimBlock.x + 2 * radius), stream>>>(dst, src, (unsigned)width, (unsigned)height,
                                                                            radius);
        break;
      case 3:
        gaussianBlur1DRGBA210SSKernelInterior<unrolledGaussianKernel3>
            <<<dimGrid, dimBlock, 16 * (dimBlock.x + 2 * radius), stream>>>(dst, src, (unsigned)width, (unsigned)height,
                                                                            radius);
        break;
      case 4:
        gaussianBlur1DRGBA210SSKernelInterior<unrolledGaussianKernel4>
            <<<dimGrid, dimBlock, 16 * (dimBlock.x + 2 * radius), stream>>>(dst, src, (unsigned)width, (unsigned)height,
                                                                            radius);
        break;
      case 5:
        gaussianBlur1DRGBA210SSKernelInterior<unrolledGaussianKernel5>
            <<<dimGrid, dimBlock, 16 * (dimBlock.x + 2 * radius), stream>>>(dst, src, (unsigned)width, (unsigned)height,
                                                                            radius);
        break;
      case 6:
        gaussianBlur1DRGBA210SSKernelInterior<unrolledGaussianKernel6>
            <<<dimGrid, dimBlock, 16 * (dimBlock.x + 2 * radius), stream>>>(dst, src, (unsigned)width, (unsigned)height,
                                                                            radius);
        break;
      default:
        assert(false);
        break;
    }
  }
  // There are exactly radius pixels on each border (left and right) + radius pixels before and after them.
  assert(4 * radius <= 32);
  dim3 dimBlock(4 * radius, 1, 1);
  dim3 dimGrid(1, (unsigned)height, 1);
  if (wrap) {
    switch (radius) {
      case 1:
        gaussianBlur1DRGBA210SSKernelWrap<unrolledGaussianKernel1>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 2:
        gaussianBlur1DRGBA210SSKernelWrap<unrolledGaussianKernel2>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 3:
        gaussianBlur1DRGBA210SSKernelWrap<unrolledGaussianKernel3>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 4:
        gaussianBlur1DRGBA210SSKernelWrap<unrolledGaussianKernel4>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 5:
        gaussianBlur1DRGBA210SSKernelWrap<unrolledGaussianKernel5>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 6:
        gaussianBlur1DRGBA210SSKernelWrap<unrolledGaussianKernel6>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      default:
        assert(false);
        break;
    }
  } else {
    switch (radius) {
      case 1:
        gaussianBlur1DRGBA210SSKernelNoWrap<unrolledGaussianKernel1>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 2:
        gaussianBlur1DRGBA210SSKernelNoWrap<unrolledGaussianKernel2>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 3:
        gaussianBlur1DRGBA210SSKernelNoWrap<unrolledGaussianKernel3>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 4:
        gaussianBlur1DRGBA210SSKernelNoWrap<unrolledGaussianKernel4>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 5:
        gaussianBlur1DRGBA210SSKernelNoWrap<unrolledGaussianKernel5>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      case 6:
        gaussianBlur1DRGBA210SSKernelNoWrap<unrolledGaussianKernel6>
            <<<dimGrid, dimBlock, 16 * dimBlock.x, stream>>>(dst, src, (unsigned)width, (unsigned)height, radius);
        break;
      default:
        assert(false);
        break;
    }
  }
  return CUDA_STATUS;
}

Status gaussianBlur2DRGBA210SS(uint32_t* buf, uint32_t* work, std::size_t width, std::size_t height, unsigned radius,
                               bool wrap, GPU::Stream stream) {
  // Vertical pass, never wraps.
  PROPAGATE_FAILURE_STATUS(gaussianBlur1DRGBA210SS(work, buf, width, height, radius, false, stream));
  // transpose
  PROPAGATE_FAILURE_STATUS(transpose(buf, work, width, height, stream));
  PROPAGATE_FAILURE_STATUS(gaussianBlur1DRGBA210SS(work, buf, height, width, radius, wrap, stream));
  return transpose(buf, work, height, width, stream);
}

// TODO_GPU_DEPRECATE
// only used in test currently
Status gaussianBlur2D(GPU::Buffer<unsigned char> buf, GPU::Buffer<unsigned char> work, std::size_t width,
                      std::size_t height, unsigned radius, unsigned passes, bool wrap, unsigned blockSize,
                      GPU::Stream stream) {
  // Avoid copy: force even passes
  assert((passes & 1) == 0);
  for (unsigned i = 0; i < passes / 2; ++i) {
    PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(work, buf.as_const(), width, height, radius, blockSize, stream));
    PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(buf, work.as_const(), width, height, radius, blockSize, stream));
  }
  // transpose
  PROPAGATE_FAILURE_STATUS(transpose(work.get().raw(), buf.as_const().get().raw(), width, height, stream));
  for (unsigned i = 0; i < passes / 2; ++i) {
    if (wrap) {
      PROPAGATE_FAILURE_STATUS(boxBlur1DWrap(buf, work.as_const(), height, width, radius, blockSize, stream));
      PROPAGATE_FAILURE_STATUS(boxBlur1DWrap(work, buf.as_const(), height, width, radius, blockSize, stream));
    } else {
      PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(buf, work.as_const(), height, width, radius, blockSize, stream));
      PROPAGATE_FAILURE_STATUS(boxBlur1DNoWrap(work, buf.as_const(), height, width, radius, blockSize, stream));
    }
  }
  return transpose(buf.get().raw(), work.as_const().get().raw(), height, width, stream);
}

Status gaussianBlur2DRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, GPU::Buffer<uint32_t> work,
                          std::size_t width, std::size_t height, unsigned /*radius*/, unsigned /*passes*/, bool wrap,
                          GPU::Stream stream) {
  uint32_t* h_Kernel = (uint32_t*)malloc((2 * KERNEL_RADIUS + 1) * sizeof(uint32_t));
  h_Kernel[0] = 1;
  h_Kernel[1] = 4;
  h_Kernel[2] = 6;
  h_Kernel[3] = 4;
  h_Kernel[4] = 1;
  setConvolutionKernel(h_Kernel);

  {
    dim3 blocks((unsigned)Cuda::ceilDiv(width, ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X),
                (unsigned)Cuda::ceilDiv(height, ROWS_BLOCKDIM_Y));
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
    if (wrap) {
      convolutionRowsKernel<true><<<blocks, threads, 0, stream.get()>>>(
          work.get().raw(), src.get().raw(), (unsigned)width, (unsigned)height, (unsigned)width);
    } else {
      convolutionRowsKernel<false><<<blocks, threads, 0, stream.get()>>>(
          work.get().raw(), src.get().raw(), (unsigned)width, (unsigned)height, (unsigned)width);
    }
  }

  {
    dim3 blocks((unsigned)Cuda::ceilDiv(width, COLUMNS_BLOCKDIM_X),
                (unsigned)Cuda::ceilDiv(height, COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
    dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

    convolutionColumnsKernel<<<blocks, threads, 0, stream.get()>>>(dst.get().raw(), work.get().raw(), (unsigned)width,
                                                                   (unsigned)height, (unsigned)width);
  }

  free(h_Kernel);

  return CUDA_STATUS;
}

}  // namespace Image
}  // namespace VideoStitch
