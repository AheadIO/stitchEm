// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "image/unpack.hpp"
#include "colorArrayDevice.hpp"

#include "backend/cuda/deviceBuffer.hpp"
#include "backend/cuda/deviceBuffer2D.hpp"
#include "backend/cuda/surface.hpp"
#include "backend/cuda/deviceStream.hpp"

#include "cuda/util.hpp"

#include "unpackKernel.cu"

#include <hip/hip_runtime.h>
#include <cassert>

const unsigned int CudaBlockSize = 16;

namespace VideoStitch {
namespace Image {

// ---------------- Convert RGBA -> other colorspace --------------------------

Status unpackRGB(GPU::Buffer2D& dst, const GPU::Buffer<const uint32_t>& array, std::size_t width, std::size_t height,
                 GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  unpackKernelRGB<<<dimGrid, dimBlock, 0, s.get()>>>(dst.get().raw(), (unsigned)dst.getPitch(), array.get(),
                                                     (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackRGB(GPU::Buffer2D& dst, const GPU::Surface& surf, std::size_t width, std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  unpackSourceKernelRGB<<<dimGrid, dimBlock, 0, s.get()>>>(dst.get().raw(), (unsigned)dst.getPitch(),
                                                           surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackRGBA(GPU::Buffer2D& dst, const GPU::Buffer<const uint32_t>& array, std::size_t /*width*/,
                  std::size_t /*height*/, GPU::Stream s) {
  return CUDA_ERROR(hipMemcpy2DAsync(dst.get().raw(), (unsigned)dst.getPitch(), array.get(), dst.getWidth(),
                                      dst.getWidth(), dst.getHeight(), hipMemcpyDeviceToDevice, s.get()));
}

Status unpackRGBA(GPU::Buffer2D& dst, const GPU::Surface& surf, std::size_t width, std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  unpackSourceKernelRGBA<<<dimGrid, dimBlock, 0, s.get()>>>(
      (uint32_t*)dst.get().raw(), (unsigned)dst.getPitch() / sizeof(uint32_t),  // pitch is in bytes
      surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackF32C1(GPU::Buffer2D& dst, const GPU::Buffer<const uint32_t>& array, std::size_t /*width*/,
                   std::size_t /*height*/, GPU::Stream s) {
  return CUDA_ERROR(hipMemcpy2DAsync(dst.get().raw(), (unsigned)dst.getPitch(), array.get(), dst.getWidth(),
                                      dst.getWidth(), dst.getHeight(), hipMemcpyDeviceToDevice, s.get()));
}

Status unpackF32C1(GPU::Buffer2D& dst, const GPU::Surface& surf, std::size_t width, std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  unpackSourceKernelF32C1<<<dimGrid, dimBlock, 0, s.get()>>>(
      (float*)dst.get().raw(), (unsigned)dst.getPitch() / sizeof(float),  // pitch is in bytes
      surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackGrayscale16(GPU::Buffer2D& /* dst */, const GPU::Buffer<const uint32_t>& /* input */, size_t /* width*/,
                         size_t /* height */, GPU::Stream /* s */) {
  // TODO
  return {Origin::GPU, ErrType::UnsupportedAction,
          "Color space conversion for Grayscale16 not implemented from buffer"};
}

Status unpackGrayscale16(GPU::Buffer2D& dst, const GPU::Surface& surf, size_t width, size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  unpackSourceKernelGrayscale16<<<dimGrid, dimBlock, 0, s.get()>>>(
      (uint16_t*)dst.get().raw(), (unsigned)dst.getPitch() / sizeof(uint16_t),  // pitch is in bytes
      surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackDepth(GPU::Buffer2D& yDst, GPU::Buffer2D& uDst, GPU::Buffer2D& vDst,
                   const GPU::Buffer<const uint32_t>& array, std::size_t width, std::size_t height, GPU::Stream s) {
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv((height + 1) / 2, dimBlock.y), 1);
  unpackKernelDepth<<<dimGrid, dimBlock, 0, s.get()>>>(
      yDst.get().raw(), (unsigned)yDst.getPitch(), uDst.get().raw(), (unsigned)uDst.getPitch(), vDst.get().raw(),
      (unsigned)vDst.getPitch(), (float*)array.get().raw(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackDepth(GPU::Buffer2D& yDst, GPU::Buffer2D& uDst, GPU::Buffer2D& vDst, const GPU::Surface& surf,
                   std::size_t width, std::size_t height, GPU::Stream s) {
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv((height + 1) / 2, dimBlock.y), 1);
  unpackSourceKernelDepth<<<dimGrid, dimBlock, 0, s.get()>>>(
      yDst.get().raw(), (unsigned)yDst.getPitch(), uDst.get().raw(), (unsigned)uDst.getPitch(), vDst.get().raw(),
      (unsigned)vDst.getPitch(), surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackYV12(GPU::Buffer2D& yDst, GPU::Buffer2D& uDst, GPU::Buffer2D& vDst,
                  const GPU::Buffer<const uint32_t>& array, std::size_t width, std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv((height + 1) / 2, dimBlock.y), 1);
  unpackKernelYV12<<<dimGrid, dimBlock, 0, s.get()>>>(
      yDst.get().raw(), (unsigned)yDst.getPitch(), uDst.get().raw(), (unsigned)uDst.getPitch(), vDst.get().raw(),
      (unsigned)vDst.getPitch(), array.get(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackYV12(GPU::Buffer2D& yDst, GPU::Buffer2D& uDst, GPU::Buffer2D& vDst, const GPU::Surface& surf,
                  std::size_t width, std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv((height + 1) / 2, dimBlock.y), 1);
  unpackSourceKernelYV12<<<dimGrid, dimBlock, 0, s.get()>>>(
      yDst.get().raw(), (unsigned)yDst.getPitch(), uDst.get().raw(), (unsigned)uDst.getPitch(), vDst.get().raw(),
      (unsigned)vDst.getPitch(), surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackNV12(GPU::Buffer2D& yDst, GPU::Buffer2D& uvDst, const GPU::Surface& surf, std::size_t width,
                  std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv((height + 1) / 2, dimBlock.y), 1);
  unpackSourceKernelNV12<<<dimGrid, dimBlock, 0, s.get()>>>(yDst.get().raw(), (unsigned)yDst.getPitch(),
                                                            uvDst.get().raw(), (unsigned)uvDst.getPitch(),
                                                            surf.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackNV12(GPU::Buffer2D& yDst, GPU::Buffer2D& uvDst, const GPU::Buffer<const uint32_t>& array,
                  std::size_t width, std::size_t height, GPU::Stream s) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x),
                     (unsigned)Cuda::ceilDiv((height + 1) / 2, dimBlock.y), 1);
  unpackKernelNV12<<<dimGrid, dimBlock, 0, s.get()>>>(yDst.get().raw(), (unsigned)yDst.getPitch(), uvDst.get().raw(),
                                                      (unsigned)uvDst.getPitch(), array.get(), (unsigned)width,
                                                      (unsigned)height);
  return CUDA_STATUS;
}

Status unpackYUY2(GPU::Buffer2D& dst, const GPU::Buffer<const uint32_t>& src, std::size_t width, std::size_t height,
                  GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y),
                     1);
  unpackYUY2Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().raw(), (unsigned)dst.getPitch(), src.get(),
                                                           (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackYUY2(GPU::Buffer2D&, const GPU::Surface&, std::size_t, std::size_t, GPU::Stream) {
  return Status{Origin::GPU, ErrType::ImplementationError, "Unpacking not implemented from Surface"};
}

Status unpackUYVY(GPU::Buffer2D& dst, const GPU::Buffer<const uint32_t>& src, std::size_t width, std::size_t height,
                  GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y),
                     1);
  unpackUYVYKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().raw(), (unsigned)dst.getPitch(), src.get(),
                                                           (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status unpackUYVY(GPU::Buffer2D&, const GPU::Surface&, std::size_t, std::size_t, GPU::Stream) {
  return Status{Origin::GPU, ErrType::ImplementationError, "Unpacking not implemented from Surface"};
}

Status convertGrayscale(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                        std::size_t height, GPU::Stream stream) {
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width, dimBlock2D.x), (unsigned)Cuda::ceilDiv(height, dimBlock2D.y), 1);
  convertKernelGrayscale<<<dimGrid2D, dimBlock2D, 0, stream.get()>>>(dst.get(), src.get(), (unsigned)width,
                                                                     (unsigned)height);
  return CUDA_STATUS;
}

Status unpackYUV422P10(GPU::Buffer2D& yDst, GPU::Buffer2D& uDst, GPU::Buffer2D& vDst,
                       const GPU::Buffer<const uint32_t>& src, std::size_t width, std::size_t height,
                       GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv((width + 1) / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y),
                     1);
  unpackYUV422P10Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      reinterpret_cast<uint16_t*>(yDst.get().raw()), (unsigned)yDst.getPitch() / 2,
      reinterpret_cast<uint16_t*>(uDst.get().raw()), (unsigned)uDst.getPitch() / 2,
      reinterpret_cast<uint16_t*>(vDst.get().raw()), (unsigned)vDst.getPitch() / 2, src.get(), (unsigned)width,
      (unsigned)height);
  return CUDA_STATUS;
}

Status unpackYUV422P10(GPU::Buffer2D&, GPU::Buffer2D&, GPU::Buffer2D&, const GPU::Surface&, std::size_t, std::size_t,
                       GPU::Stream) {
  return Status{Origin::GPU, ErrType::ImplementationError, "Unpacking not implemented from Surface"};
}

Status unpackGrayscale(GPU::Buffer2D& dst, const GPU::Surface& src, std::size_t width, std::size_t height,
                       GPU::Stream stream) {
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width, dimBlock2D.x), (unsigned)Cuda::ceilDiv(height, dimBlock2D.y), 1);
  unpackKernelGrayscale<<<dimGrid2D, dimBlock2D, 0, stream.get()>>>(
      dst.get().raw(), (unsigned)dst.getPitch(), src.get().surface(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

// ---------------- Convert other colorspace -> RGBA --------------------------

Status convertRGBToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width, std::size_t height,
                        GPU::Stream stream) {
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width, dimBlock2D.x), (unsigned)Cuda::ceilDiv(height, dimBlock2D.y), 1);
  convertRGBToRGBAKernel<<<dimGrid2D, dimBlock2D, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                     (unsigned)height);
  return CUDA_STATUS;
}

Status convertRGB210ToRGBA(GPU::Surface& dst, GPU::Buffer<const uint32_t> src, std::size_t width, std::size_t height,
                           GPU::Stream stream) {
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width, dimBlock2D.x), (unsigned)Cuda::ceilDiv(height, dimBlock2D.y), 1);
  convertRGB210ToRGBAKernel<<<dimGrid2D, dimBlock2D, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                        (unsigned)height);
  return CUDA_STATUS;
}

Status convertBGRToRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                        std::size_t height, GPU::Stream stream) {
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width, dimBlock2D.x), (unsigned)Cuda::ceilDiv(height, dimBlock2D.y), 1);
  convertBGRToRGBAKernel<<<dimGrid2D, dimBlock2D, 0, stream.get()>>>(dst.get(), src.get(), (unsigned)width,
                                                                     (unsigned)height);
  return CUDA_STATUS;
}

Status convertBGRUToRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                         std::size_t height, GPU::Stream stream) {
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width / 2, dimBlock2D.x),
                       (unsigned)Cuda::ceilDiv(height / 2, dimBlock2D.y), 1);
  convertBGRUToRGBAKernel<<<dimGrid2D, dimBlock2D, 0, stream.get()>>>(dst.get(), src.get(), (unsigned)width,
                                                                      (unsigned)height);
  return CUDA_STATUS;
}

Status convertBayerRGGBToRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, GPU::Stream stream) {
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width / 2, dimBlock2D.x),
                       (unsigned)Cuda::ceilDiv(height / 2, dimBlock2D.y), 1);
  convertBayerRGGBToRGBAKernel<<<dimGrid2D, dimBlock2D, sizeof(uint32_t) * (dimBlock2D.x + 1) * (dimBlock2D.y + 1),
                                 stream.get()>>>(dst.get(), src.get(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status convertBayerBGGRToRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, GPU::Stream stream) {
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width / 2, dimBlock2D.x),
                       (unsigned)Cuda::ceilDiv(height / 2, dimBlock2D.y), 1);
  convertBayerBGGRToRGBAKernel<<<dimGrid2D, dimBlock2D, sizeof(uint32_t) * (dimBlock2D.x + 1) * (dimBlock2D.y + 1),
                                 stream.get()>>>(dst.get(), src.get(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status convertBayerGRBGToRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, GPU::Stream stream) {
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width / 2, dimBlock2D.x),
                       (unsigned)Cuda::ceilDiv(height / 2, dimBlock2D.y), 1);
  convertBayerGRBGToRGBAKernel<<<dimGrid2D, dimBlock2D, sizeof(uint32_t) * (dimBlock2D.x + 1) * (dimBlock2D.y + 1),
                                 stream.get()>>>(dst.get(), src.get(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status convertBayerGBRGToRGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, GPU::Stream stream) {
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimBlock2D(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid2D((unsigned)Cuda::ceilDiv(width / 2, dimBlock2D.x),
                       (unsigned)Cuda::ceilDiv(height / 2, dimBlock2D.y), 1);
  convertBayerGBRGToRGBAKernel<<<dimGrid2D, dimBlock2D, sizeof(uint32_t) * (dimBlock2D.x + 1) * (dimBlock2D.y + 1),
                                 stream.get()>>>(dst.get(), src.get(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status convertUYVYToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width, std::size_t height,
                         GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  convertUYVYToRGBAKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                  (unsigned)height);
  return CUDA_STATUS;
}

Status convertYUV422P10ToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, GPU::Stream stream) {
  assert(!(width & 1));
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  convertYUV422P10ToRGBAKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      dst.get().surface(), src.as<const uint16_t>().get(), (unsigned)width, (unsigned)height);
  return CUDA_STATUS;
}

Status convertYUY2ToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width, std::size_t height,
                         GPU::Stream stream) {
  const dim3 dimBlock(16, 16, 1);
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  convertYUY2ToRGBAKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                  (unsigned)height);
  return CUDA_STATUS;
}

Status convertYV12ToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width, std::size_t height,
                         GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height / 2, dimBlock.y),
                     1);
  convertYV12ToRGBAKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                  (unsigned)height);
  return CUDA_STATUS;
}

Status convertNV12ToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width, std::size_t height,
                         GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height / 2, dimBlock.y),
                     1);
  convertNV12ToRGBAKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                  (unsigned)height);
  return CUDA_STATUS;
}

Status convertYUV420ToMono(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                           std::size_t height, GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  assert(!(width & 1));
  assert(!(height & 1));
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width / 2, dimBlock.x), (unsigned)Cuda::ceilDiv(height / 2, dimBlock.y),
                     1);
  unpackMonoKernelYUV420P<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get(), src.get(), (unsigned)width,
                                                                  (unsigned)height);
  return CUDA_STATUS;
}

Status convertGrayscaleToRGBA(GPU::Surface& dst, GPU::Buffer<const unsigned char> src, std::size_t width,
                              std::size_t height, GPU::Stream stream) {
  const dim3 dimBlock(CudaBlockSize, CudaBlockSize, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  convertGrayscaleKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get().surface(), src.get(), (unsigned)width,
                                                                 (unsigned)height);
  return CUDA_STATUS;
}
}  // namespace Image
}  // namespace VideoStitch
