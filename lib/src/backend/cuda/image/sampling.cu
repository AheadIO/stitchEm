#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/image/sampling.hpp"

#include "../deviceBuffer.hpp"
#include "../deviceStream.hpp"
#include "../surface.hpp"
#include "../gpuKernelDef.h"

#include "backend/common/vectorOps.hpp"
#include "cuda/util.hpp"
#include "image/kernels/sharedUtils.hpp"
#include "backend/cuda/core1/kernels/samplingKernel.cu"
#include <hip/hip_runtime.h>
#include <cassert>

#include "backend/common/image/sampling.gpu"

namespace VideoStitch {
namespace Image {

// ------------------- Subsampling

template <>
Status subsample22(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t srcWidth,
                   std::size_t srcHeight, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  std::size_t dstWidth = (srcWidth + 1) / 2;
  std::size_t dstHeight = (srcHeight + 1) / 2;
  // interior
  {
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22RegularKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(), (unsigned)srcWidth,
                                                               (unsigned)srcHeight, (unsigned)dstWidth,
                                                               (unsigned)dstHeight);
  }
  // right boundary
  if (srcWidth & 1) {
    dim3 dimBlock(1, 256, 1);
    dim3 dimGrid(1, (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22RightBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                     (unsigned)srcWidth, (unsigned)srcHeight,
                                                                     (unsigned)dstWidth, (unsigned)dstHeight);
  }
  // bottom boundary
  if (srcHeight & 1) {
    dim3 dimBlock(256, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), 1, 1);
    subsample22BottomBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                      (unsigned)srcWidth, (unsigned)srcHeight,
                                                                      (unsigned)dstWidth, (unsigned)dstHeight);
  }
  if ((srcWidth & 1) && (srcHeight & 1)) {
    // simple copy
    unsigned char* dstPtr = (unsigned char*)dst.get().raw();
    unsigned char* srcPtr = (unsigned char*)src.get().raw();
    return CUDA_ERROR(hipMemcpyAsync(dstPtr + dstWidth * dstHeight - 1, srcPtr + srcHeight * srcWidth - 1,
                                      sizeof(unsigned char), hipMemcpyDeviceToDevice, stream));
  }
  return CUDA_STATUS;
}

template <>
Status subsample22(GPU::Buffer<float2> dst, GPU::Buffer<const float2> src, std::size_t srcWidth, std::size_t srcHeight,
                   GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  std::size_t dstWidth = (srcWidth + 1) / 2;
  std::size_t dstHeight = (srcHeight + 1) / 2;
  // interior
  {
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22RegularKernelFloat2<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                     (unsigned)srcWidth, (unsigned)srcHeight,
                                                                     (unsigned)dstWidth, (unsigned)dstHeight);
  }
  // right boundary
  if (srcWidth & 1) {
    dim3 dimBlock(1, 256, 1);
    dim3 dimGrid(1, (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22RightBoundaryKernelFloat2<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                           (unsigned)srcWidth, (unsigned)srcHeight,
                                                                           (unsigned)dstWidth, (unsigned)dstHeight);
  }
  // bottom boundary
  if (srcHeight & 1) {
    dim3 dimBlock(256, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), 1, 1);
    subsample22BottomBoundaryKernelFloat2<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                            (unsigned)srcWidth, (unsigned)srcHeight,
                                                                            (unsigned)dstWidth, (unsigned)dstHeight);
  }
  if ((srcWidth & 1) && (srcHeight & 1)) {
    // simple copy
    float2* dstPtr = (float2*)dst.get().raw();
    float2* srcPtr = (float2*)src.get().raw();
    return CUDA_ERROR(hipMemcpyAsync(dstPtr + dstWidth * dstHeight - 1, srcPtr + srcHeight * srcWidth - 1,
                                      sizeof(float2), hipMemcpyDeviceToDevice, stream));
  }
  return CUDA_STATUS;
}

template <>
Status subsample22Mask(GPU::Buffer<float2> dst, GPU::Buffer<uint32_t> dstMask, GPU::Buffer<const float2> src,
                       GPU::Buffer<const uint32_t> srcMask, std::size_t srcWidth, std::size_t srcHeight,
                       unsigned blockSize, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  std::size_t dstWidth = (srcWidth + 1) / 2;
  std::size_t dstHeight = (srcHeight + 1) / 2;
  // interior
  {
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22MaskRegularKernel<<<dimGrid, dimBlock, 0, stream>>>(
        dst.get().raw(), dstMask.get().raw(), src.get().raw(), srcMask.get().raw(), (unsigned)srcWidth,
        (unsigned)srcHeight, (unsigned)dstWidth, (unsigned)dstHeight);
  }
  // right boundary
  if (srcWidth & 1) {
    dim3 dimBlock(1, blockSize * blockSize, 1);
    dim3 dimGrid(1, (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22MaskRightBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(
        dst.get().raw(), dstMask.get().raw(), src.get().raw(), srcMask.get().raw(), (unsigned)srcWidth,
        (unsigned)srcHeight, (unsigned)dstWidth, (unsigned)dstHeight);
  }
  // bottom boundary
  if (srcHeight & 1) {
    dim3 dimBlock(blockSize * blockSize, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), 1, 1);
    subsample22MaskBottomBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(
        dst.get().raw(), dstMask.get().raw(), src.get().raw(), srcMask.get().raw(), (unsigned)srcWidth,
        (unsigned)srcHeight, (unsigned)dstWidth, (unsigned)dstHeight);
  }
  if ((srcWidth & 1) && (srcHeight & 1)) {
    // simple copy
    float2* dstPtr = (float2*)dst.get().raw();
    float2* srcPtr = (float2*)src.get().raw();
    FAIL_RETURN(CUDA_ERROR(hipMemcpyAsync(dstPtr + dstWidth * dstHeight - 1, srcPtr + srcHeight * srcWidth - 1,
                                           sizeof(float2), hipMemcpyDeviceToDevice, stream)));

    uint32_t* dstMaskPtr = (uint32_t*)dstMask.get().raw();
    uint32_t* srcMaskPtr = (uint32_t*)srcMask.get().raw();
    FAIL_RETURN(CUDA_ERROR(hipMemcpyAsync(dstMaskPtr + dstWidth * dstHeight - 1, srcMaskPtr + srcHeight * srcWidth - 1,
                                           sizeof(uint32_t), hipMemcpyDeviceToDevice, stream)));
  }
  return CUDA_STATUS;
}

template <typename T>
Status subsample22Nearest(GPU::Buffer<T> dst, GPU::Buffer<const T> src, std::size_t srcWidth, std::size_t srcHeight,
                          unsigned blockSize, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  std::size_t dstWidth = (srcWidth + 1) / 2;
  std::size_t dstHeight = (srcHeight + 1) / 2;
  // interior
  {
    dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22NearestRegularKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                      (unsigned)srcWidth, (unsigned)srcHeight,
                                                                      (unsigned)dstWidth, (unsigned)dstHeight);
  }
  // right boundary
  if (srcWidth & 1) {
    dim3 dimBlock(1, blockSize * blockSize, 1);
    dim3 dimGrid(1, (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22NearestRightBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                            (unsigned)srcWidth, (unsigned)srcHeight,
                                                                            (unsigned)dstWidth, (unsigned)dstHeight);
  }
  if (srcHeight & 1) {
    dim3 dimBlock(blockSize * blockSize, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), 1, 1);
    subsample22NearestBottomBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get().raw(), src.get().raw(),
                                                                             (unsigned)srcWidth, (unsigned)srcHeight,
                                                                             (unsigned)dstWidth, (unsigned)dstHeight);
  }
  if ((srcWidth & 1) && (srcHeight & 1)) {
    // simple copy
    T* dstPtr = (T*)dst.get().raw();
    T* srcPtr = (T*)src.get().raw();
    return CUDA_ERROR(hipMemcpyAsync(dstPtr + dstWidth * dstHeight - 1, srcPtr + srcHeight * srcWidth - 1, sizeof(T),
                                      hipMemcpyDeviceToDevice, stream));
  }
  return CUDA_STATUS;
}

// template
// Status subsample22Nearest(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t srcWidth,
// std::size_t srcHeight, unsigned blockSize, GPU::Stream stream); template Status
// subsample22Nearest(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t srcWidth,
// std::size_t srcHeight, unsigned blockSize, GPU::Stream stream); template Status subsample22Nearest(GPU::Buffer<float>
// dst, GPU::Buffer<const float> src, std::size_t srcWidth, std::size_t srcHeight, unsigned blockSize, GPU::Stream
// stream);

Status subsample22RGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t srcWidth,
                       std::size_t srcHeight, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();

  std::size_t dstWidth = (srcWidth + 1) / 2;
  std::size_t dstHeight = (srcHeight + 1) / 2;
  // interior
  {
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22RGBARegularKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get(), src.get(), (unsigned)srcWidth,
                                                                   (unsigned)srcHeight, (unsigned)dstWidth);
  }
  // right boundary
  if (srcWidth & 1) {
    dim3 dimBlock(1, 256, 1);
    dim3 dimGrid(1, (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
    subsample22RGBARightBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get(), src.get(), (unsigned)srcWidth,
                                                                         (unsigned)srcHeight, (unsigned)dstWidth);
  }
  if (srcHeight & 1) {
    dim3 dimBlock(256, 1, 1);
    dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), 1, 1);
    subsample22RGBABottomBoundaryKernel<<<dimGrid, dimBlock, 0, stream>>>(dst.get(), src.get(), (unsigned)srcWidth,
                                                                          (unsigned)srcHeight, (unsigned)dstWidth);
  }
  if ((srcWidth & 1) && (srcHeight & 1)) {
    // simple copy
    uint32_t* dstPtr = (uint32_t*)dst.get().raw();
    uint32_t* srcPtr = (uint32_t*)src.get().raw();
    return CUDA_ERROR(hipMemcpyAsync(dstPtr + dstWidth * dstHeight - 1, srcPtr + srcHeight * srcWidth - 1, 4,
                                      hipMemcpyDeviceToDevice, stream));
  }
  return CUDA_STATUS;
}

// ------------------ Upsampling

/**
 * Upsample @src by a factor of two on each dimension and put it in into @dst.
 * @dst has size (@dstWidth * @dstHeight), @dst has size ((@dstWidth + 1)/2 * (@dstHeight + 1)/2).
 * This is more complex than subsampling since we need to interpolate at the same time.
 * We use shared memory to share reads to global memory between threads.
 * In addition, we make sure that memory accesses are coalesced.
 * To avoid divergence in the regular case, there are two kernels: one that applies inside
 * the image, and one that applies to boundaries.
 * The alpha is taken to be solid if at least one sample is solid.
 */

/**
  // Bilinear interpolation.
  //                           +=======+=======+=======+
  //                           |       |       |       |
  //                           |   A   |   B   |   C   |
  //  |       |       |        |       |       |       |
  //  +=======+=======+=       +=======+===+===+=======+
  //  |       |       |        |       | a | b |       |
  //  |   D   |   E   |        |   D   +---+---+   F   |
  //  |       |       |        |       | c | d |       |
  //  +=======+=======+=  =>   +=======+===+===+=======+
  //  |       |       |        |       |       |       |
  //  |   G   |   H   |        |   G   |   H   |   I   |
  //  |       |       |        |       |       |       |
  //  +=======+=======+=       +=======+=======+=======+
  //
  // The current thread loads source pixel E, then computes interpolated values for a, b, c, d:
  //    a = 1 / 16 * A + 3 / 16 * [D + B] + 9 / 16 * E
  //    b = 1 / 16 * C + 3 / 16 * [B + F] + 9 / 16 * E
  //    c = 1 / 16 * G + 3 / 16 * [D + H] + 9 / 16 * E
  //    d = 1 / 16 * I + 3 / 16 * [F + H] + 9 / 16 * E
*/

struct BilinearInterpolationRGB210 {
  typedef uint32_t Type;

  static inline __device__ uint32_t interpolate(uint32_t a, uint32_t b, uint32_t c, uint32_t d) {
    // see above
    const int32_t alphaA = RGB210::a(a);
    const int32_t alphaB = RGB210::a(b);
    const int32_t alphaC = RGB210::a(c);
    const int32_t alphaD = RGB210::a(d);
    const int32_t divisor = 9 * alphaA + 3 * (alphaB + alphaC) + alphaD;
    return RGB210::pack(
        (alphaA * 9 * RGB210::r(a) + 3 * (alphaB * RGB210::r(b) + alphaC * RGB210::r(c)) + alphaD * RGB210::r(d)) /
            divisor,
        (alphaA * 9 * RGB210::g(a) + 3 * (alphaB * RGB210::g(b) + alphaC * RGB210::g(c)) + alphaD * RGB210::g(d)) /
            divisor,
        (alphaA * 9 * RGB210::b(a) + 3 * (alphaB * RGB210::b(b) + alphaC * RGB210::b(c)) + alphaD * RGB210::b(d)) /
            divisor,
        divisor > 0);
  }
};

struct BilinearInterpolationRGBA {
  typedef uint32_t Type;

  static inline __device__ uint32_t interpolate(uint32_t a, uint32_t b, uint32_t c, uint32_t d) {
    // see above
    const uint32_t alphaA = !!RGBA::a(a);
    const uint32_t alphaB = !!RGBA::a(b);
    const uint32_t alphaC = !!RGBA::a(c);
    const uint32_t alphaD = !!RGBA::a(d);
    const uint32_t divisor = 9 * alphaA + 3 * (alphaB + alphaC) + alphaD;
    if (divisor) {
      return RGBASolid::pack(
          (alphaA * 9 * RGBA::r(a) + 3 * (alphaB * RGBA::r(b) + alphaC * RGBA::r(c)) + alphaD * RGBA::r(d)) / divisor,
          (alphaA * 9 * RGBA::g(a) + 3 * (alphaB * RGBA::g(b) + alphaC * RGBA::g(c)) + alphaD * RGBA::g(d)) / divisor,
          (alphaA * 9 * RGBA::b(a) + 3 * (alphaB * RGBA::b(b) + alphaC * RGBA::b(c)) + alphaD * RGBA::b(d)) / divisor,
          0xff);
    } else {
      return 0;
    }
  }
};

template <typename T>
struct BilinearInterpolation {
  typedef T Type;

  static inline __device__ T interpolate(T a, T b, T c, T d) {
    // see above
    return (T)(9.0f / 16.0f * a + 3.0f / 16.0f * (b + c) + 1.0f / 16.0f * d);
  }
};

Status upsample22RGBA210(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t dstWidth,
                         std::size_t dstHeight, bool wrap, GPU::Stream stream) {
  const unsigned srcWidth = ((unsigned)dstWidth + 1) / 2;
  const unsigned srcHeight = ((unsigned)dstHeight + 1) / 2;
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(srcWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(srcHeight, dimBlock.y), 1);
  if (wrap) {
    upsample22Kernel<HWrapBoundary<uint32_t>, BilinearInterpolationRGB210>
        <<<dimGrid, dimBlock, (16 + 2) * (16 + 2) * 4, stream.get()>>>(dst.get(), src.get(), (unsigned)dstWidth,
                                                                       (unsigned)dstHeight, srcWidth, srcHeight);
  } else {
    upsample22Kernel<ExtendBoundary<uint32_t>, BilinearInterpolationRGB210>
        <<<dimGrid, dimBlock, (16 + 2) * (16 + 2) * 4, stream.get()>>>(dst.get(), src.get(), (unsigned)dstWidth,
                                                                       (unsigned)dstHeight, srcWidth, srcHeight);
  }
  return CUDA_STATUS;
}

Status upsample22RGBA(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t dstWidth,
                      std::size_t dstHeight, bool wrap, GPU::Stream stream) {
  const unsigned srcWidth = ((unsigned)dstWidth + 1) / 2;
  const unsigned srcHeight = ((unsigned)dstHeight + 1) / 2;
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(srcWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(srcHeight, dimBlock.y), 1);
  if (wrap) {
    upsample22Kernel<HWrapBoundary<uint32_t>, BilinearInterpolationRGBA>
        <<<dimGrid, dimBlock, (16 + 2) * (16 + 2) * 4, stream.get()>>>(dst.get(), src.get(), (unsigned)dstWidth,
                                                                       (unsigned)dstHeight, srcWidth, srcHeight);
  } else {
    upsample22Kernel<ExtendBoundary<uint32_t>, BilinearInterpolationRGBA>
        <<<dimGrid, dimBlock, (16 + 2) * (16 + 2) * 4, stream.get()>>>(dst.get(), src.get(), (unsigned)dstWidth,
                                                                       (unsigned)dstHeight, srcWidth, srcHeight);
  }
  return CUDA_STATUS;
}

template <typename T>
Status upsample22(GPU::Buffer<T> dst, GPU::Buffer<const T> src, std::size_t dstWidth, std::size_t dstHeight, bool wrap,
                  GPU::Stream stream) {
  const unsigned srcWidth = ((unsigned)dstWidth + 1) / 2;
  const unsigned srcHeight = ((unsigned)dstHeight + 1) / 2;
  const dim3 dimBlock(16, 16, 1);
  const dim3 dimGrid((unsigned)Cuda::ceilDiv(srcWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(srcHeight, dimBlock.y), 1);
  if (wrap) {
    upsample22Kernel<HWrapBoundary<T>, BilinearInterpolation<T>><<<dimGrid, dimBlock, 0, stream.get()>>>(
        dst.get(), src.get(), (unsigned)dstWidth, (unsigned)dstHeight, srcWidth, srcHeight);
  } else {
    upsample22Kernel<ExtendBoundary<T>, BilinearInterpolation<T>><<<dimGrid, dimBlock, 0, stream.get()>>>(
        dst.get(), src.get(), (unsigned)dstWidth, (unsigned)dstHeight, srcWidth, srcHeight);
  }
  return CUDA_STATUS;
}

template Status upsample22(GPU::Buffer<uint32_t> dst, GPU::Buffer<const uint32_t> src, std::size_t dstWidth,
                           std::size_t dstHeight, bool wrap, GPU::Stream stream);
template Status upsample22(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t dstWidth,
                           std::size_t dstHeight, bool wrap, GPU::Stream stream);
template Status upsample22(GPU::Buffer<float> dst, GPU::Buffer<const float> src, std::size_t dstWidth,
                           std::size_t dstHeight, bool wrap, GPU::Stream stream);
template Status upsample22(GPU::Buffer<float2> dst, GPU::Buffer<const float2> src, std::size_t dstWidth,
                           std::size_t dstHeight, bool wrap, GPU::Stream stream);

// ---------------- Masks sampling

Status subsampleMask22(GPU::Buffer<unsigned char> dst, GPU::Buffer<const unsigned char> src, std::size_t srcWidth,
                       std::size_t srcHeight, unsigned blockSize, GPU::Stream stream) {
  std::size_t dstWidth = (srcWidth + 1) / 2;
  std::size_t dstHeight = (srcHeight + 1) / 2;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(dstWidth, dimBlock.x), (unsigned)Cuda::ceilDiv(dstHeight, dimBlock.y), 1);
  subsampleMask22Kernel<<<dimGrid, dimBlock, 0, stream.get()>>>(dst.get(), src.get(), (unsigned)srcWidth,
                                                                (unsigned)srcHeight, (unsigned)dstWidth);
  return CUDA_STATUS;
}

}  // namespace Image
}  // namespace VideoStitch
