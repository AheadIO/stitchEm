#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "core1/panoRemapper.hpp"

#include "core/rect.hpp"
#include "core/transformGeoParams.hpp"

#include "backend/common/imageOps.hpp"

#include "backend/cuda/deviceBuffer.hpp"
#include "backend/cuda/deviceStream.hpp"
#include "backend/cuda/surface.hpp"

#include "cuda/error.hpp"
#include "cuda/util.hpp"

#include "gpu/core1/transform.hpp"
#include "gpu/buffer.hpp"
#include "gpu/memcpy.hpp"
#include "gpu/allocator.hpp"

#include "libvideostitch/panoDef.hpp"

#include "backend/cuda/core/transformStack.cu"
#include "core/kernels/withinStack.cu"

namespace VideoStitch {
namespace Core {

template <Convert2D3DFnT toSphere, class OutputCropper>
__global__ void remapKernel(uint32_t* g_odata, hipTextureObject_t remapTex, int panoWidth, int panoHeight,
                            const float2 inPanoScale, const float2 outPanoScale, const vsfloat3x3 R) {
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < panoWidth && y < panoHeight) {
    if (OutputCropper::isPanoPointVisible(x, y, panoWidth, panoHeight)) {
      float2 uv = make_float2((float)x, (float)y);

      /**
       * The transformations are applied relative to the center of the panorama image
       */
      uv.x -= (panoWidth - 1) / 2.0f;
      uv.y -= (panoHeight - 1) / 2.0f;

      /**
       * Apply transform stack
       */
      uv.x /= outPanoScale.x;
      uv.y /= outPanoScale.y;

      float3 pt = toSphere(uv);

      pt = rotateSphere(pt, R);

      uv = SphereToErect(pt);

      uv.x *= inPanoScale.x;
      uv.y *= inPanoScale.y;

      /**
       * See notes in warp kernel
       * compensate fetching offset with hipFilterModeLinear by adding 0.5f
       * https://stackoverflow.com/questions/10643790/texture-memory-tex2d-basics
       */
      uv.x += panoWidth / 2.0f;
      uv.y += panoHeight / 2.0f;

      float4 px = tex2D<float4>(remapTex, uv.x, uv.y);
      g_odata[y * panoWidth + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                     __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
    } else {
      g_odata[y * panoWidth + x] = 0;
    }
  }
}

__global__ void remapCubemapKernel(uint32_t* __restrict__ xPositive, uint32_t* __restrict__ xNegative,
                                   uint32_t* __restrict__ yPositive, uint32_t* __restrict__ yNegative,
                                   uint32_t* __restrict__ zPositive, uint32_t* __restrict__ zNegative, int panoWidth,
                                   int panoHeight, hipTextureObject_t remapTex, int faceDim, const float2 panoScale,
                                   const vsfloat3x3 R) {
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < faceDim && y < faceDim) {
    /* compensate fetching offset with hipFilterModeLinear by adding 0.5f */
    float2 uv = make_float2(x + 0.5f, y + 0.5f);
    uv = (uv / faceDim) * 2.f - make_float2(1.f, 1.f);

    float3 pt;
    for (unsigned int face = 0; face < 6; face++) {
      // Layer 0 is positive X face
      if (face == 0) {
        pt.x = 1;
        pt.y = -uv.y;
        pt.z = -uv.x;
      }
      // Layer 1 is negative X face
      else if (face == 1) {
        pt.x = -1;
        pt.y = -uv.y;
        pt.z = uv.x;
      }
      // Layer 2 is positive Y face
      else if (face == 2) {
        pt.x = uv.x;
        pt.y = 1;
        pt.z = uv.y;
      }
      // Layer 3 is negative Y face
      else if (face == 3) {
        pt.x = uv.x;
        pt.y = -1;
        pt.z = -uv.y;
      }
      // Layer 4 is positive Z face
      else if (face == 4) {
        pt.x = uv.x;
        pt.y = -uv.y;
        pt.z = 1;
      }
      // Layer 5 is negative Z face
      else if (face == 5) {
        pt.x = -uv.x;
        pt.y = -uv.y;
        pt.z = -1;
      }

      pt = rotateSphere(pt, R);

      float2 xy = SphereToErect(pt);

      xy *= panoScale;

      /**
       * See notes in warp kernel
       */
      xy.x += panoWidth / 2.0f;
      xy.y += panoHeight / 2.0f;

      float4 px = tex2D<float4>(remapTex, xy.x, xy.y);
      if (face == 0) {
        xPositive[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 1) {
        xNegative[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 2) {
        yPositive[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 3) {
        yNegative[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 4) {
        zPositive[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 5) {
        zNegative[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      }
    }
  }
}

template <bool equiangular>
__global__ void rotateCubemapKernel(uint32_t* __restrict__ xPositive, uint32_t* __restrict__ xNegative,
                                    uint32_t* __restrict__ yPositive, uint32_t* __restrict__ yNegative,
                                    uint32_t* __restrict__ zPositive, uint32_t* __restrict__ zNegative, int faceDim,
                                    hipTextureObject_t remapTex, const vsfloat3x3 R) {
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < faceDim && y < faceDim) {
    float2 uv = make_float2((float)x, (float)y);
    uv = (uv / faceDim) * 2.f - make_float2(1.f, 1.f);

    if (equiangular) {
      uv.x = tanf_vs(uv.x * PI_F_VS / 4.);
      uv.y = tanf_vs(uv.y * PI_F_VS / 4.);
    }

    float3 pt;
    for (unsigned int face = 0; face < 6; face++) {
      // Layer 0 is positive X face
      if (face == 0) {
        pt.x = 1;
        pt.y = -uv.y;
        pt.z = -uv.x;
      }
      // Layer 1 is negative X face
      else if (face == 1) {
        pt.x = -1;
        pt.y = -uv.y;
        pt.z = uv.x;
      }
      // Layer 2 is positive Y face
      else if (face == 2) {
        pt.x = uv.x;
        pt.y = 1;
        pt.z = uv.y;
      }
      // Layer 3 is negative Y face
      else if (face == 3) {
        pt.x = uv.x;
        pt.y = -1;
        pt.z = -uv.y;
      }
      // Layer 4 is positive Z face
      else if (face == 4) {
        pt.x = uv.x;
        pt.y = -uv.y;
        pt.z = 1;
      }
      // Layer 5 is negative Z face
      else if (face == 5) {
        pt.x = -uv.x;
        pt.y = -uv.y;
        pt.z = -1;
      }

      pt = rotateSphere(pt, R);

      if (equiangular) {
        // first normalize with Chebyshev distance to project back on the cube
        float cheb = fmaxf(abs(pt.x), abs(pt.y));
        cheb = fmaxf(cheb, abs(pt.z));
        pt /= cheb;
        // then reinflate the cube
        pt.x = 4. / PI_F_VS * atanf_vs(pt.x);
        pt.y = 4. / PI_F_VS * atanf_vs(pt.y);
        pt.z = 4. / PI_F_VS * atanf_vs(pt.z);
      }

      float4 px = texCubemap<float4>(remapTex, pt.x, pt.y, pt.z);
      if (face == 0) {
        xPositive[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 1) {
        xNegative[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 2) {
        yPositive[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 3) {
        yNegative[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 4) {
        zPositive[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      } else if (face == 5) {
        zNegative[y * faceDim + x] = Image::RGBA::pack(__float2uint_rn(px.x * 255.), __float2uint_rn(px.y * 255.),
                                                       __float2uint_rn(px.z * 255.), __float2uint_rn(px.w * 255.));
      }
    }
  }
}

Status rotateCubemap(const PanoDefinition& pano, GPU::CubemapSurface& cubemapSurface, GPU::Buffer<uint32_t> xPosPbo,
                     GPU::Buffer<uint32_t> xNegPbo, GPU::Buffer<uint32_t> yPosPbo, GPU::Buffer<uint32_t> yNegPbo,
                     GPU::Buffer<uint32_t> zPosPbo, GPU::Buffer<uint32_t> zNegPbo, const Matrix33<double>& perspective,
                     bool equiangular, GPU::Stream stream) {
  vsfloat3x3 rotation;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      rotation.values[i][j] = (float)perspective(i, j);
    }
  }

  dim3 block(16, 16, 1);
  dim3 grid((unsigned)Cuda::ceilDiv(pano.getLength(), block.x), (unsigned)Cuda::ceilDiv(pano.getLength(), block.y), 1);

  if (equiangular) {
    rotateCubemapKernel<true><<<grid, block, 0, stream.get()>>>(
        xPosPbo.get(), xNegPbo.get(), yPosPbo.get(), yNegPbo.get(), zPosPbo.get(), zNegPbo.get(), (int)pano.getLength(),
        cubemapSurface.get().texture(), rotation);
  } else {
    rotateCubemapKernel<false><<<grid, block, 0, stream.get()>>>(
        xPosPbo.get(), xNegPbo.get(), yPosPbo.get(), yNegPbo.get(), zPosPbo.get(), zNegPbo.get(), (int)pano.getLength(),
        cubemapSurface.get().texture(), rotation);
  }

  return CUDA_STATUS;
}

__device__ float3 positiveX(float2& uv) {
  float3 pt;
  pt.x = 1;
  pt.y = -uv.y;
  pt.z = uv.x;
  return pt;
}
__device__ float3 negativeX(float2& uv) {
  float3 pt;
  pt.x = -1;
  pt.y = -uv.y;
  pt.z = -uv.x;
  return pt;
}
__device__ float3 positiveY(float2& uv) {
  float3 pt;
  pt.x = uv.x;
  pt.y = 1;
  pt.z = -uv.y;
  return pt;
}
__device__ float3 negativeY(float2& uv) {
  float3 pt;
  pt.x = uv.x;
  pt.y = -1;
  pt.z = uv.y;
  return pt;
}
__device__ float3 positiveZ(float2& uv) {
  float3 pt;
  pt.x = uv.x;
  pt.y = -uv.y;
  pt.z = -1;
  return pt;
}
__device__ float3 negativeZ(float2& uv) {
  float3 pt;
  pt.x = -uv.x;
  pt.y = -uv.y;
  pt.z = 1;
  return pt;
}

template <float3 (*project)(float2&), bool equiangular>
__global__ void remapMaskFace(unsigned char* __restrict__ face, int dstOffsetX, int dstOffsetY, int bbWidth,
                              int bbHeight, int panoWidth, int panoHeight, hipTextureObject_t remapTex, int srcOffsetX,
                              int srcOffsetY, int faceDim, const float2 panoScale) {
  unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < bbWidth && y < bbHeight) {
    /* compensate fetching offset with hipFilterModeLinear by adding 0.5f */
    float2 uv = make_float2(x + dstOffsetX + 0.5f, y + dstOffsetY + 0.5f);
    uv = (uv / faceDim) * 2.f - make_float2(1.f, 1.f);

    if (equiangular) {
      uv.x = tanf_vs(uv.x * PI_F_VS / 4.);
      uv.y = tanf_vs(uv.y * PI_F_VS / 4.);
    }

    float3 pt = project(uv);

    float2 xy = SphereToErect(pt);

    xy *= panoScale;

    /**
     * See notes in warp kernel
     */
    xy.x += panoWidth / 2.0f;
    xy.y += panoHeight / 2.0f;

    xy.x -= srcOffsetX;
    xy.y -= srcOffsetY;
    if (xy.x < 0.) {
      xy.x += panoWidth;
    }

    float px = tex2D<float>(remapTex, xy.x, xy.y);
    face[y * bbWidth + x] = __float2uint_rn(px * 255.);
  }
}

Status reprojectAlphaToCubemap(int panoWidth, int panoHeight, int faceLength, GPU::Surface& alphaSurface,
                               Rect equirectBB, GPU::Buffer<unsigned char> xPosAlpha, Rect xPosBB,
                               GPU::Buffer<unsigned char> xNegAlpha, Rect xNegBB, GPU::Buffer<unsigned char> yPosAlpha,
                               Rect yPosBB, GPU::Buffer<unsigned char> yNegAlpha, Rect yNegBB,
                               GPU::Buffer<unsigned char> zPosAlpha, Rect zPosBB, GPU::Buffer<unsigned char> zNegAlpha,
                               Rect zNegBB, bool equiangular, GPU::Stream stream) {
  dim3 block(16, 16, 1);
  float2 panoScale = {TransformGeoParams::computePanoScale(PanoProjection::Equirectangular, panoWidth, 360.f),
                      2 * TransformGeoParams::computePanoScale(PanoProjection::Equirectangular, panoHeight, 360.f)};

  if (!xPosBB.empty()) {
    dim3 gridXPos((unsigned)Cuda::ceilDiv(xPosBB.getWidth(), block.x),
                  (unsigned)Cuda::ceilDiv(xPosBB.getHeight(), block.y), 1);
    if (equiangular) {
      remapMaskFace<positiveX, true><<<gridXPos, block, 0, stream.get()>>>(
          xPosAlpha.get().raw(), (unsigned)xPosBB.left(), (unsigned)xPosBB.top(), (unsigned)xPosBB.getWidth(),
          (unsigned)xPosBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    } else {
      remapMaskFace<positiveX, false><<<gridXPos, block, 0, stream.get()>>>(
          xPosAlpha.get().raw(), (unsigned)xPosBB.left(), (unsigned)xPosBB.top(), (unsigned)xPosBB.getWidth(),
          (unsigned)xPosBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    }
  }
  if (!xNegBB.empty()) {
    dim3 gridXNeg((unsigned)Cuda::ceilDiv(xNegBB.getWidth(), block.x),
                  (unsigned)Cuda::ceilDiv(xNegBB.getHeight(), block.y), 1);
    if (equiangular) {
      remapMaskFace<negativeX, true><<<gridXNeg, block, 0, stream.get()>>>(
          xNegAlpha.get().raw(), (unsigned)xNegBB.left(), (unsigned)xNegBB.top(), (unsigned)xNegBB.getWidth(),
          (unsigned)xNegBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    } else {
      remapMaskFace<negativeX, false><<<gridXNeg, block, 0, stream.get()>>>(
          xNegAlpha.get().raw(), (unsigned)xNegBB.left(), (unsigned)xNegBB.top(), (unsigned)xNegBB.getWidth(),
          (unsigned)xNegBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    }
  }
  if (!yPosBB.empty()) {
    dim3 gridYPos((unsigned)Cuda::ceilDiv(yPosBB.getWidth(), block.x),
                  (unsigned)Cuda::ceilDiv(yPosBB.getHeight(), block.y), 1);
    if (equiangular) {
      remapMaskFace<positiveY, true><<<gridYPos, block, 0, stream.get()>>>(
          yPosAlpha.get().raw(), (unsigned)yPosBB.left(), (unsigned)yPosBB.top(), (unsigned)yPosBB.getWidth(),
          (unsigned)yPosBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    } else {
      remapMaskFace<positiveY, false><<<gridYPos, block, 0, stream.get()>>>(
          yPosAlpha.get().raw(), (unsigned)yPosBB.left(), (unsigned)yPosBB.top(), (unsigned)yPosBB.getWidth(),
          (unsigned)yPosBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    }
  }
  if (!yNegBB.empty()) {
    dim3 gridYNeg((unsigned)Cuda::ceilDiv(yNegBB.getWidth(), block.x),
                  (unsigned)Cuda::ceilDiv(yNegBB.getHeight(), block.y), 1);
    if (equiangular) {
      remapMaskFace<negativeY, true><<<gridYNeg, block, 0, stream.get()>>>(
          yNegAlpha.get().raw(), (unsigned)yNegBB.left(), (unsigned)yNegBB.top(), (unsigned)yNegBB.getWidth(),
          (unsigned)yNegBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    } else {
      remapMaskFace<negativeY, false><<<gridYNeg, block, 0, stream.get()>>>(
          yNegAlpha.get().raw(), (unsigned)yNegBB.left(), (unsigned)yNegBB.top(), (unsigned)yNegBB.getWidth(),
          (unsigned)yNegBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    }
  }
  if (!zPosBB.empty()) {
    dim3 gridZPos((unsigned)Cuda::ceilDiv(zPosBB.getWidth(), block.x),
                  (unsigned)Cuda::ceilDiv(zPosBB.getHeight(), block.y), 1);
    if (equiangular) {
      remapMaskFace<positiveZ, true><<<gridZPos, block, 0, stream.get()>>>(
          zPosAlpha.get().raw(), (unsigned)zPosBB.left(), (unsigned)zPosBB.top(), (unsigned)zPosBB.getWidth(),
          (unsigned)zPosBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    } else {
      remapMaskFace<positiveZ, false><<<gridZPos, block, 0, stream.get()>>>(
          zPosAlpha.get().raw(), (unsigned)zPosBB.left(), (unsigned)zPosBB.top(), (unsigned)zPosBB.getWidth(),
          (unsigned)zPosBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    }
  }
  if (!zNegBB.empty()) {
    dim3 gridZNeg((unsigned)Cuda::ceilDiv(zNegBB.getWidth(), block.x),
                  (unsigned)Cuda::ceilDiv(zNegBB.getHeight(), block.y), 1);
    if (equiangular) {
      remapMaskFace<negativeZ, true><<<gridZNeg, block, 0, stream.get()>>>(
          zNegAlpha.get().raw(), (unsigned)zNegBB.left(), (unsigned)zNegBB.top(), (unsigned)zNegBB.getWidth(),
          (unsigned)zNegBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    } else {
      remapMaskFace<negativeZ, false><<<gridZNeg, block, 0, stream.get()>>>(
          zNegAlpha.get().raw(), (unsigned)zNegBB.left(), (unsigned)zNegBB.top(), (unsigned)zNegBB.getWidth(),
          (unsigned)zNegBB.getHeight(), (unsigned)panoWidth, (unsigned)panoHeight, alphaSurface.get().texture(),
          (unsigned)equirectBB.left(), (unsigned)equirectBB.top(), (unsigned)faceLength, panoScale);
    }
  }
  return CUDA_STATUS;
}

template <Convert2D3DFnT toSphere, class OutputCropper>
Status reprojectPanorama(GPU::Buffer<uint32_t> pbo, float2 dstScale, GPU::Surface& tex, float2 srcScale, unsigned width,
                         unsigned height, const Matrix33<double>& perspective, GPU::Stream stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);

  vsfloat3x3 rotation;
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      rotation.values[i][j] = (float)perspective(i, j);
    }
  }

  remapKernel<toSphere, OutputCropper><<<dimGrid, dimBlock, 0, stream.get()>>>(pbo.get(), tex.get().texture(), width,
                                                                               height, srcScale, dstScale, rotation);

  return CUDA_STATUS;
}

Status reprojectRectilinear(GPU::Buffer<uint32_t> pbo, float2 outScale, GPU::Surface& tex, float2 inScale,
                            unsigned width, unsigned height, const Matrix33<double>& perspective, GPU::Stream stream) {
  return reprojectPanorama<RectToSphere, OutputRectCropper>(pbo, outScale, tex, inScale, width, height, perspective,
                                                            stream);
}
Status reprojectEquirectangular(GPU::Buffer<uint32_t> pbo, float2 outScale, GPU::Surface& tex, float2 inScale,
                                unsigned width, unsigned height, const Matrix33<double>& perspective,
                                GPU::Stream stream) {
  return reprojectPanorama<ErectToSphere, OutputRectCropper>(pbo, outScale, tex, inScale, width, height, perspective,
                                                             stream);
}
Status reprojectFullFrameFisheye(GPU::Buffer<uint32_t> pbo, float2 outScale, GPU::Surface& tex, float2 inScale,
                                 unsigned width, unsigned height, const Matrix33<double>& perspective,
                                 GPU::Stream stream) {
  return reprojectPanorama<FisheyeToSphere, OutputRectCropper>(pbo, outScale, tex, inScale, width, height, perspective,
                                                               stream);
}
Status reprojectCircularFisheye(GPU::Buffer<uint32_t> pbo, float2 outScale, GPU::Surface& tex, float2 inScale,
                                unsigned width, unsigned height, const Matrix33<double>& perspective,
                                GPU::Stream stream) {
  return reprojectPanorama<FisheyeToSphere, OutputCircleCropper>(pbo, outScale, tex, inScale, width, height,
                                                                 perspective, stream);
}
Status reprojectStereographic(GPU::Buffer<uint32_t> pbo, float2 outScale, GPU::Surface& tex, float2 inScale,
                              unsigned width, unsigned height, const Matrix33<double>& perspective,
                              GPU::Stream stream) {
  return reprojectPanorama<StereoToSphere, OutputRectCropper>(pbo, outScale, tex, inScale, width, height, perspective,
                                                              stream);
}

}  // namespace Core
}  // namespace VideoStitch
