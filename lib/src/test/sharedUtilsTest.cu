#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm
//
// Basic input unpacking tests.

#include "testing.hpp"
#include "util.hpp"
#include "../image/kernels/sharedUtils.hpp"

#include <memory>

namespace VideoStitch {
namespace Testing {

template <typename T, unsigned left, unsigned right, unsigned top, unsigned bottom, typename Getter>
__global__ void dumpSharedKernel(T* __restrict__ sharedDst, const unsigned sharedWidth, const unsigned sharedHeight,
                                 const T* __restrict__ src, unsigned srcWidth, unsigned srcHeight, unsigned srcOffsetX,
                                 unsigned srcOffsetY) {
  Image::loadToSharedMemory<T, left, right, top, bottom, Getter>(sharedDst, sharedWidth, sharedHeight, src, srcWidth,
                                                                 srcHeight, srcOffsetX, srcOffsetY);
  __syncthreads();
}

template <typename T, unsigned left, unsigned right, unsigned top, unsigned bottom>
void groundTruthExtend(std::vector<T>& sharedDst, const int sharedWidth, const int sharedHeight,
                       const std::vector<T>& src, int srcWidth, int srcHeight, int srcOffsetX, int srcOffsetY) {
  const int realSharedWidth = sharedWidth + left + right;
  const int realSharedHeight = sharedHeight + top + bottom;
  sharedDst.clear();
  // Top rows:
  for (int sharedY = 0; sharedY < realSharedHeight; ++sharedY) {
    int srcY = srcOffsetY + sharedY - top;
    if (srcY < 0) {
      srcY = 0;
    }
    if (srcY >= srcHeight) {
      srcY = srcHeight - 1;
    }
    for (int sharedX = 0; sharedX < realSharedWidth; ++sharedX) {
      int srcX = srcOffsetX + sharedX - left;
      if (srcX < 0) {
        srcX = 0;
      }
      if (srcX >= srcWidth) {
        srcX = srcWidth - 1;
      }
      sharedDst.push_back(src[srcWidth * srcY + srcX]);
    }
  }
}

template <typename T, unsigned left, unsigned right, unsigned top, unsigned bottom>
void groundTruthZero(std::vector<T>& sharedDst, const int sharedWidth, const int sharedHeight,
                     const std::vector<T>& src, int srcWidth, int srcHeight, int srcOffsetX, int srcOffsetY) {
  const int realSharedWidth = sharedWidth + left + right;
  const int realSharedHeight = sharedHeight + top + bottom;
  sharedDst.clear();
  // Top rows:
  for (int sharedY = 0; sharedY < realSharedHeight; ++sharedY) {
    const int srcY = srcOffsetY + sharedY - top;
    for (int sharedX = 0; sharedX < realSharedWidth; ++sharedX) {
      const int srcX = srcOffsetX + sharedX - left;
      if (srcY < 0 || srcY >= srcHeight || srcX < 0 || srcX >= srcWidth) {
        sharedDst.push_back(0);
      } else {
        sharedDst.push_back(src[srcWidth * srcY + srcX]);
      }
    }
  }
}

template <typename T, unsigned left, unsigned right, unsigned top, unsigned bottom, int blockWidth, int blockHeight>
void runTest(const int width, const int height, const std::vector<T>& input, const int sharedWidth,
             const int sharedHeight, const int srcOffsetX, const int srcOffsetY, bool extend) {
  // std::cout << "sharedWidth=" << sharedWidth << " sharedHeight=" << sharedHeight << " srcOffsetX=" << srcOffsetX << "
  // srcOffsetY=" << srcOffsetY << std::endl;

  std::vector<T> output;
  {
    DeviceBuffer<T> inputBuffer(width, height);
    inputBuffer.fill(input);
    const dim3 dimBlock2D(blockWidth, blockHeight, 1);
    const dim3 dimGrid2D(1, 1, 1);
    DeviceBuffer<T> outputBuffer(sharedWidth + left + right, sharedHeight + top + bottom);
    outputBuffer.fill((T)99);
    if (extend) {
      dumpSharedKernel<T, left, right, top, bottom, Image::ExtendBoundary<T>><<<dimGrid2D, dimBlock2D, 0, 0>>>(
          outputBuffer.ptr(), sharedWidth, sharedHeight, inputBuffer.ptr(), width, height, srcOffsetX, srcOffsetY);
    } else {
      dumpSharedKernel<T, left, right, top, bottom, Image::ZeroBoundary<T>><<<dimGrid2D, dimBlock2D, 0, 0>>>(
          outputBuffer.ptr(), sharedWidth, sharedHeight, inputBuffer.ptr(), width, height, srcOffsetX, srcOffsetY);
    }

    outputBuffer.readback(output);
  }

  std::vector<T> groundTruthOutput;
  if (extend) {
    groundTruthExtend<T, left, right, top, bottom>(groundTruthOutput, sharedWidth, sharedHeight, input, width, height,
                                                   srcOffsetX, srcOffsetY);
  } else {
    groundTruthZero<T, left, right, top, bottom>(groundTruthOutput, sharedWidth, sharedHeight, input, width, height,
                                                 srcOffsetX, srcOffsetY);
  }

  /*for (int y = 0; y < sharedHeight + top + bottom; ++y) {
    for (int x = 0; x < sharedWidth + left + right; ++x) {
      std::cout << groundTruthOutput[(sharedWidth + left + right) * y + x]<< " ";
    }
    std::cout << std::endl;
  }

  for (int y = 0; y < sharedHeight + top + bottom; ++y) {
    for (int x = 0; x < sharedWidth + left + right; ++x) {
      std::cout << output[(sharedWidth + left + right) * y + x]<< " ";
    }
    std::cout << std::endl;
  }*/
  ENSURE_2D_ARRAY_EQ(groundTruthOutput.data(), output.data(), sharedWidth + left + right, sharedHeight + top + bottom);
}

template <typename T, unsigned left, unsigned right, unsigned top, unsigned bottom>
void testSharedUtils(const int width, const int height, bool extend) {
  std::vector<T> input;
  for (int i = 0; i < width * height; ++i) {
    input.push_back((T)i);
  }

  for (int sharedWidth = 1; sharedWidth < width; ++sharedWidth) {
    for (int sharedHeight = 1; sharedHeight < height; ++sharedHeight) {
      for (int srcOffsetX = 0; srcOffsetX < width; ++srcOffsetX) {
        for (int srcOffsetY = 0; srcOffsetY < height; ++srcOffsetY) {
          runTest<T, left, right, top, bottom, 4, 4>(width, height, input, sharedWidth, sharedHeight, srcOffsetX,
                                                     srcOffsetY, extend);
        }
      }
    }
  }
}
}  // namespace Testing
}  // namespace VideoStitch

int main() {
  hipSetDevice(0);
  VideoStitch::Testing::testSharedUtils<int, 1u, 1u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<int, 2u, 1u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<int, 1u, 2u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<int, 1u, 1u, 2u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<int, 1u, 1u, 1u, 2u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<char, 1u, 1u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<char, 2u, 1u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<char, 1u, 2u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<char, 1u, 1u, 2u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<char, 1u, 1u, 1u, 2u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<uint16_t, 1u, 1u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<uint16_t, 2u, 1u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<uint16_t, 1u, 2u, 1u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<uint16_t, 1u, 1u, 2u, 1u>(6, 5, true);
  VideoStitch::Testing::testSharedUtils<uint16_t, 1u, 1u, 1u, 2u>(6, 5, true);

  VideoStitch::Testing::testSharedUtils<int, 1u, 1u, 1u, 1u>(6, 5, false);

  {
    const int width = 512;
    const int height = 512;
    std::vector<float> input;
    for (int i = 0; i < width * height; ++i) {
      input.push_back((float)i);
    }
    VideoStitch::Testing::runTest<float, 1u, 1u, 1u, 1u, 16, 16>(width, height, input, 16, 16, 110, 11, true);
  }
  hipDeviceReset();
  return 0;
}
