#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "parallax/mergerPair.hpp"

#include "backend/cuda/deviceBuffer.hpp"
#include "backend/cuda/deviceStream.hpp"
#include "cuda/error.hpp"
#include "cuda/util.hpp"
#include "core/rect.hpp"
#include "parallax/flowConstant.hpp"

#include <string.h>

namespace VideoStitch {
namespace Core {

#define TILE_X 16
#define TILE_Y 16

__global__ void pairMappingMaskKernel(const int wrapWidth, const int input0OffsetX, const int input0OffsetY,
                                      const int input0Width, const int input0Height, const float2* input0CoordBuffer,
                                      const int input1OffsetX, const int input1OffsetY, const int input1Width,
                                      const int input1Height, const float2* input1CoordBuffer, const int outputOffsetX,
                                      const int outputOffsetY, const int outputWidth, const int outputHeight,
                                      uint32_t* outputMask) {
  // calculate normalized texture coordinates
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < outputWidth && y < outputHeight) {
    outputMask[y * outputWidth + x] = 0;

    // Put the first mask in
    const int input0X = (x + outputOffsetX - input0OffsetX + wrapWidth) % wrapWidth;
    const int input0Y = (y + outputOffsetY - input0OffsetY);
    if (input0X >= 0 && input0X < input0Width && input0Y >= 0 && input0Y < input0Height) {
      float2 coord0 = input0CoordBuffer[input0Y * input0Width + input0X];
      if (coord0.x != INVALID_FLOW_VALUE && coord0.y != INVALID_FLOW_VALUE) {
        outputMask[y * outputWidth + x] += 1 << 1;
      }
    }

    // Put the second mask in
    const int input1X = (x + outputOffsetX - input1OffsetX + wrapWidth) % wrapWidth;
    const int input1Y = (y + outputOffsetY - input1OffsetY);
    if (input1X >= 0 && input1X < input1Width && input1Y >= 0 && input1Y < input1Height) {
      float2 coord1 = input1CoordBuffer[input1Y * input1Width + input1X];
      if (coord1.x != INVALID_FLOW_VALUE && coord1.y != INVALID_FLOW_VALUE) {
        outputMask[y * outputWidth + x] += 1 << 2;
      }
    }
  }
}

Status MergerPair::setupPairMappingMask(GPU::Buffer<uint32_t> devMask, GPU::Stream gpuStream) const {
  const Rect rect0 = getBoundingPanoRect(0);
  const Rect rect1 = getBoundingPanoRect(1);
  Rect iRect = getBoundingPanosIRect();
  dim3 dimBlock(TILE_X, TILE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(iRect.getWidth(), dimBlock.x),
               (unsigned)Cuda::ceilDiv(iRect.getHeight(), dimBlock.y), 1);

  hipStream_t stream = gpuStream.get();

  pairMappingMaskKernel<<<dimGrid, dimBlock, 0, stream>>>(
      (int)wrapWidth, (int)rect0.left(), (int)rect0.top(), (int)rect0.getWidth(), (int)rect0.getHeight(),
      panoToInputSpaceCoordMapping0.borrow_const().get(), (int)rect1.left(), (int)rect1.top(), (int)rect1.getWidth(),
      (int)rect1.getHeight(), panoToInputSpaceCoordMapping1.borrow_const().get(), (int)iRect.left(), (int)iRect.top(),
      (int)iRect.getWidth(), (int)iRect.getHeight(), devMask.get());

  return CUDA_STATUS;
}

}  // namespace Core
}  // namespace VideoStitch
