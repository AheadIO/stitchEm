#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "parallax/simpleFlow.hpp"

#include "./kernels/patchDifferenceFunction.cu"

#include "backend/common/vectorOps.hpp"

#include "backend/cuda/deviceBuffer.hpp"
#include "backend/cuda/deviceStream.hpp"
#include "backend/cuda/core1/kernels/samplingKernel.cu"
#include "gpu/image/sampling.hpp"
#include "gpu/image/imageOps.hpp"
#include "gpu/image/blur.hpp"
#include "gpu/stream.hpp"
#include "cuda/error.hpp"
#include "cuda/util.hpp"
#include "util/imageProcessingGPUUtils.hpp"

namespace VideoStitch {
namespace Core {

#define TILE_WIDTH 16
#define CUDABLOCKSIZE 512
#define SIMPLEFLOW_KERNEL_BLOCK_SIZE_X 16
#define SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y 8

__global__ void forwardFlowKernel(const int flowSize, const int windowSize, const float flowMagnitudeWeight,
                                  const float gradientWeight, const int2 size0, const int2 offset0,
                                  const uint32_t* input0, const float* gradient0, const int2 size1, const int2 offset1,
                                  const uint32_t* input1, const float* gradient1, const float2* inputFlowOffset,
                                  float2* flow, float* confidence) {
  // Check whether we need to calculate the flow
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size0.x || y >= size0.y) return;

  uint32_t v0 = input0[y * size0.x + x];
  if (Image::RGBA::a(v0) == 0) {  // If current alpha is 0, do nothing
    if (inputFlowOffset) {
      flow[y * size0.x + x] = inputFlowOffset[y * size0.x + x];
      if (confidence) {
        confidence[y * size0.x + x] = 1;
      }
      return;
    }
    flow[y * size0.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
    if (confidence) {
      confidence[y * size0.x + x] = 0;
    }
    return;
  }
  int2 coord1 = make_int2(x + offset0.x - offset1.x, y + offset0.y - offset1.y);
  if (!inRange(coord1, size1)) {
    flow[y * size0.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
    if (confidence) {
      confidence[y * size0.x + x] = 0;
    }
    return;
  }
  uint32_t v1 = input1[coord1.y * size1.x + coord1.x];
  if (Image::RGBA::a(v1) == 0) {
    flow[y * size0.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
    if (confidence) {
      confidence[y * size0.x + x] = 0;
    }
    return;
  }

  float2 flowOffset = make_float2(0, 0);
  if (inputFlowOffset) {
    flowOffset = inputFlowOffset[y * size0.x + x];
  }
  // Try to find the best forward flow here
  int2 sampleCoord = make_int2(x, y);
  float bestCost = MAX_INVALID_COST;
  float2 minFlow = make_float2(flowOffset.x, flowOffset.y);
  float totalCost = 0;
  float totalCount = 0;
  for (int i = -flowSize; i <= flowSize; i++)
    for (int j = -flowSize; j <= flowSize; j++) {
      int2 mapCoord =
          make_int2(flowOffset.x + x + i + offset0.x - offset1.x, flowOffset.y + y + j + offset0.y - offset1.y);
      if (inRange(mapCoord, size1)) {
        float cost = getCost(windowSize, gradientWeight, size0, input0, gradient0, sampleCoord, size1, input1,
                             gradient1, mapCoord) +
                     flowMagnitudeWeight * sqrtf(i * i + j * j) / sqrtf(2 * flowSize * flowSize);
        if (cost < MAX_INVALID_COST) {
          totalCost += cost;
          totalCount++;
        }
        if (cost < bestCost) {
          bestCost = cost;
          minFlow = make_float2(flowOffset.x + i, flowOffset.y + j);
        }
      }
    }

  flow[y * size0.x + x] = minFlow;

  if (confidence) {
    if (bestCost != MAX_INVALID_COST) {
      int2 mapCoord = make_int2(sampleCoord.x + minFlow.x + offset0.x - offset1.x,
                                sampleCoord.y + minFlow.y + offset0.y - offset1.y);
      confidence[y * size0.x + x] = (getCUR(windowSize, gradientWeight, size0, input0, gradient0, sampleCoord, size1,
                                            input1, gradient1, mapCoord));
      // confidence[y * size0.x + x] = totalCost / totalCount - bestCost;
    } else {
      confidence[y * size0.x + x] = 0;
    }
  }
}

__global__ void flowAgreementConfidenceKernel(const int flowSize, const int2 size0, const int2 offset0,
                                              const float2* flow0, const float* confidence0, const int2 size1,
                                              const int2 offset1, const float2* flow1, const float* confidence1,
                                              float* flowAgreementConfidence0) {
  // Check whether we need to calculate the agreement confidence
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size0.x || y >= size0.y) return;
  const float2 forwardFlow = flow0[y * size0.x + x];
  const float forwardConfidence = confidence0[y * size0.x + x];
  const int2 lookupCoord =
      make_int2(x + forwardFlow.x + offset0.x - offset1.x, y + forwardFlow.y + offset0.y - offset1.y);
  if (!inRange(lookupCoord, size1)) {
    flowAgreementConfidence0[y * size0.x + x] = 0;
    return;
  }
  const float2 backwardFlow = flow1[lookupCoord.y * size1.x + lookupCoord.x];
  const float backwardConfidence = confidence1[lookupCoord.y * size1.x + lookupCoord.x];

  // Check if forward and backward flow agree
  float normalizedAgreementLength =
      length(forwardFlow + backwardFlow) / (length(make_float2(2 * flowSize + 1, 2 * flowSize + 1)));
  flowAgreementConfidence0[y * size0.x + x] =
      powf(fmaxf(1 - normalizedAgreementLength, 0.0), 3)  // * forwardConfidence;
      * sqrtf(backwardConfidence * forwardConfidence);
}

__global__ void confidenceTransformKernel(const int width, const int height, const float threshold, const float gamma,
                                          const float clampedValue, const float* inputConfidence,
                                          float* outputConfidence) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height) return;
  float inputValue = inputConfidence[y * width + x];
  if (inputValue < threshold) {
    outputConfidence[y * width + x] = 0;
  } else {
    outputConfidence[y * width + x] = powf(inputValue, gamma);
  }
}

__device__ float getSpacialWeight(const float sigmaSpace, const float x) { return exp(-abs(sigmaSpace) * x * x); }

__global__ void confidenceAwareFlowBlurKernel(const bool extrapolation, const int2 size, const int kernelSize,
                                              const float sigmaSpace, const float sigmaImage,
                                              const float sigmaConfidence, const uint32_t* const inputImage,
                                              const float2* const inputFlow, const float* const inputConfidence,
                                              float2* const outputFlow) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size.x || y >= size.y) return;
  if (extrapolation) {
    float2 inFlow = inputFlow[y * size.x + x];
    if (inFlow.x != INVALID_FLOW_VALUE) {
      outputFlow[y * size.x + x] = inFlow;
      return;
    }
  }
  // check if the current flow is not valid, then just do nothing
  float maxDist = kernelSize * 1.4142;
  uint32_t imageColor;
  if (sigmaImage) {
    imageColor = inputImage[y * size.x + x];
  }
  if (!extrapolation) {
    if (sigmaImage > 0) {
      if (!Image::RGBA::a(imageColor)) {
        outputFlow[y * size.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
        return;
      }
    }
  }
  float sumWeight = 0;
  float2 sumContribution = make_float2(0, 0);

  if (maxDist == 0) maxDist = 1;
  for (int dx = -kernelSize; dx <= kernelSize; dx++) {
    for (int dy = -kernelSize; dy <= kernelSize; dy++) {
      // Here i came across a neighbor, what he is look like
      int2 neighborCoord = make_int2(x + dx, y + dy);
      if (!inRange(neighborCoord, size)) {
        continue;
      }
      float neighborConfidence = 1;
      if (inputConfidence) {
        neighborConfidence = inputConfidence[neighborCoord.y * size.x + neighborCoord.x];
      }
      if (neighborConfidence == 0) {
        continue;
      }

      float2 neighborflowOffset = inputFlow[neighborCoord.y * size.x + neighborCoord.x];
      if (neighborflowOffset.x == INVALID_FLOW_VALUE) {
        continue;
      }
      float weightImage = 1.0f;
      if (sigmaImage) {
        uint32_t imageColorNeighbor = inputImage[neighborCoord.y * size.x + neighborCoord.x];
        if (Image::RGBA::a(imageColorNeighbor) > 0 && Image::RGBA::a(imageColor) > 0) {
          const float sad = abs((float(Image::RGBA::r(imageColorNeighbor)) - Image::RGBA::r(imageColor)) / 255.0) +
                            abs((float(Image::RGBA::g(imageColorNeighbor)) - Image::RGBA::g(imageColor)) / 255.0) +
                            abs((float(Image::RGBA::b(imageColorNeighbor)) - Image::RGBA::b(imageColor)) / 255.0);
          weightImage = exp(-abs(sad * sad * sigmaImage));
        }
      }

      // Now calculate the distance between source and target
      float distSpace = length(make_float2(dx, dy)) / maxDist;
      float weightSpace = exp(-abs(distSpace * distSpace * sigmaSpace));
      // Now i do really look at the neighbor on the other side to see  how think is going on there
      float weight = weightSpace * weightImage * neighborConfidence;
      sumWeight += weight;
      sumContribution += weight * neighborflowOffset;
    }
  }

  // If my confidence is high, i would tend to keep mine, don't care about the neighbor's confidence
  // Here is where to set the weight
  if (sumWeight == 0) {
    outputFlow[y * size.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
  } else {
    outputFlow[y * size.x + x] = sumContribution / sumWeight;
  }
}

__global__ void flowConfidenceKernel(const int windowSize, const float gradientWeight, const int2 size0,
                                     const uint32_t* input0, const float* gradient0, const float2* inputFlow,
                                     const int2 size1, const uint32_t* input1, const float* gradient1,
                                     float* confidence) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size0.x || y >= size0.y) return;
  float2 flow = inputFlow[y * size0.x + x];
  if (flow.x == INVALID_FLOW_VALUE) {
    confidence[y * size0.x + x] = 0;
  } else {
    int2 mapCoord = make_int2(x + flow.x, y + flow.y);
    int2 sampleCoord = make_int2(x, y);
    confidence[y * size0.x + x] =
        getCUR(windowSize, gradientWeight, size0, input0, gradient0, sampleCoord, size1, input1, gradient1, mapCoord);
  }
}

Status SimpleFlow::findForwardFlow(const int flowSize, const int windowSize, const float flowMagnitudeWeight,
                                   const float gradientWeight, const int2 size0, const int2 offset0,
                                   const GPU::Buffer<const uint32_t> inputBuffer0,
                                   const GPU::Buffer<const float> inputGradientBuffer0, const int2 size1,
                                   const int2 offset1, const GPU::Buffer<const uint32_t> inputBuffer1,
                                   const GPU::Buffer<const float> inputGradientBuffer1, GPU::Buffer<float2> flow,
                                   GPU::Buffer<float> confidence, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size0.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size0.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  forwardFlowKernel<<<dimGrid, dimBlock, 0, stream>>>(flowSize, windowSize, flowMagnitudeWeight, gradientWeight, size0,
                                                      offset0, inputBuffer0.get(), inputGradientBuffer0.get(), size1,
                                                      offset1, inputBuffer1.get(), inputGradientBuffer1.get(), 0,
                                                      flow.get(), confidence.get());

  return CUDA_STATUS;
}

__global__ void offsetCostKernel(const int2 flowOffset, const int flowSize, const float flowMagnitudeWeight,
                                 const float gradientWeight, const int2 size0, const int2 offset0,
                                 const uint32_t* input0, const float* gradient0, const int2 size1, const int2 offset1,
                                 const uint32_t* input1, const float* gradient1, float2* cost) {
  // Check whether we need to calculate the flow
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size0.x || y >= size0.y) return;

  uint32_t v0 = input0[y * size0.x + x];
  if (Image::RGBA::a(v0) == 0) {  // If current alpha is 0, do nothing
    cost[y * size0.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
    return;
  }
  // Try to find the best forward flow here
  int2 sampleCoord = make_int2(x, y);
  int2 mapCoord = make_int2(flowOffset.x + x + offset0.x - offset1.x, flowOffset.y + y + offset0.y - offset1.y);
  cost[y * size0.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
  if (inRange(mapCoord, size1)) {
    float difCost =
        getCost(0, gradientWeight, size0, input0, gradient0, sampleCoord, size1, input1, gradient1, mapCoord) +
        flowMagnitudeWeight * sqrtf(flowOffset.x * flowOffset.x + flowOffset.y * flowOffset.y) /
            sqrtf(2 * flowSize * flowSize);
    cost[y * size0.x + x] = make_float2(difCost, difCost);
  }
}

Status SimpleFlow::findOffsetCost(const int2 flowOffset, const int flowSize, const float flowMagnitudeWeight,
                                  const float gradientWeight, const int2 size0, const int2 offset0,
                                  const GPU::Buffer<const uint32_t> inputBuffer0,
                                  const GPU::Buffer<const float> inputGradientBuffer0, const int2 size1,
                                  const int2 offset1, const GPU::Buffer<const uint32_t> inputBuffer1,
                                  const GPU::Buffer<const float> inputGradientBuffer1, GPU::Buffer<float2> cost,
                                  GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size0.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size0.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  offsetCostKernel<<<dimGrid, dimBlock, 0, stream>>>(
      flowOffset, flowSize, flowMagnitudeWeight, gradientWeight, size0, offset0, inputBuffer0.get(),
      inputGradientBuffer0.get(), size1, offset1, inputBuffer1.get(), inputGradientBuffer1.get(), cost.get());

  return CUDA_STATUS;
}

__global__ void updateBestCostKernel(const int2 flowOffset, const int2 size0, const float2* cost, float* bestCost,
                                     float2* bestOffset) {
  // Check whether we need to calculate the flow
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size0.x || y >= size0.y) return;
  if (flowOffset.x == 123456) {
    bestCost[y * size0.x + x] = MAX_INVALID_COST;
    bestOffset[y * size0.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
  }
  if (cost[y * size0.x + x].x >= 0 && cost[y * size0.x + x].x < bestCost[y * size0.x + x]) {
    bestCost[y * size0.x + x] = cost[y * size0.x + x].x;
    bestOffset[y * size0.x + x] = make_float2(flowOffset.x, flowOffset.y);
  }
}

Status SimpleFlow::updateBestCost(const int2 flowOffset, const int2 size0, const GPU::Buffer<const float2> cost,
                                  GPU::Buffer<float> bestCost, GPU::Buffer<float2> bestOffset, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size0.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size0.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  updateBestCostKernel<<<dimGrid, dimBlock, 0, stream>>>(flowOffset, size0, cost.get(), bestCost.get(),
                                                         bestOffset.get());
  return CUDA_STATUS;
}

Status SimpleFlow::findBackwardAndForwardFlowAgreementConfidence(
    const int flowSize, const int2 size0, const int2 offset0, const GPU::Buffer<const float2> flow0,
    const GPU::Buffer<const float> confidence0, const int2 size1, const int2 offset1,
    const GPU::Buffer<const float2> flow1, const GPU::Buffer<const float> confidence1,
    GPU::Buffer<float> flowAgreementConfidence0, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size0.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size0.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);

  flowAgreementConfidenceKernel<<<dimGrid, dimBlock, 0, stream>>>(flowSize, size0, offset0, flow0.get(),
                                                                  confidence0.get(), size1, offset1, flow1.get(),
                                                                  confidence1.get(), flowAgreementConfidence0.get());

  return CUDA_STATUS;
}

Status SimpleFlow::performConfidenceTransform(const int width, const int height, const float threshold,
                                              const float gamma, const float clampedValue,
                                              const GPU::Buffer<const float> inputConfidence,
                                              GPU::Buffer<float> outputConfidence, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(height, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  confidenceTransformKernel<<<dimGrid, dimBlock, 0, stream>>>(width, height, threshold, gamma, clampedValue,
                                                              inputConfidence.get(), outputConfidence.get());

  return CUDA_STATUS;
}

Status SimpleFlow::findConfidence(const int windowSize, const float gradientWeight, const int2 size0,
                                  const GPU::Buffer<const uint32_t> input0, const GPU::Buffer<const float> gradient0,
                                  GPU::Buffer<const float2> forwardFlow0, const int2 size1,
                                  const GPU::Buffer<const uint32_t> input1, const GPU::Buffer<const float> gradient1,
                                  GPU::Buffer<float> confidence, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size0.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size0.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  flowConfidenceKernel<<<dimGrid, dimBlock, 0, stream>>>(windowSize, gradientWeight, size0, input0.get(),
                                                         gradient0.get(), forwardFlow0.get(), size1, input1.get(),
                                                         gradient1.get(), confidence.get());

  return CUDA_STATUS;
}

Status SimpleFlow::performConfidenceAwareFlowInterpolation(const bool extrapolation, const int2 size,
                                                           const int kernelSize, const float sigmaSpace,
                                                           const float sigmaImage, const float sigmaConfidence,
                                                           const GPU::Buffer<const uint32_t> inputImage,
                                                           const GPU::Buffer<const float2> inputFlow,
                                                           const GPU::Buffer<const float> inputConfidence,
                                                           GPU::Buffer<float2> outputFlow, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  if (inputConfidence.wasAllocated()) {
    confidenceAwareFlowBlurKernel<<<dimGrid, dimBlock, 0, stream>>>(
        extrapolation, size, kernelSize, sigmaSpace, sigmaImage, sigmaConfidence, inputImage.get(), inputFlow.get(),
        inputConfidence.get(), outputFlow.get());
  } else {
    confidenceAwareFlowBlurKernel<<<dimGrid, dimBlock, 0, stream>>>(extrapolation, size, kernelSize, sigmaSpace,
                                                                    sigmaImage, sigmaConfidence, inputImage.get(),
                                                                    inputFlow.get(), nullptr, outputFlow.get());
  }
  return CUDA_STATUS;
}

__global__ void temporalAwareFlowBlurKernel(const bool extrapolation, const int frameId, const int frameCount,
                                            const int2 size, const int kernelSize, const float sigmaSpace,
                                            const float sigmaImage, const float sigmaTime, const float* const frames,
                                            const uint32_t* const inputImages, const float2* const inputFlows,
                                            const float* const inputConfidences, float2* const outputFlow) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= size.x || y >= size.y) return;
  int baseOffset = frameId * size.x * size.y;
  if (extrapolation) {
    float2 inFlow = inputFlows[baseOffset + y * size.x + x];
    if (inFlow.x != INVALID_FLOW_VALUE) {
      outputFlow[y * size.x + x] = inFlow;
      return;
    }
  }
  // check if the current flow is not valid, then just do nothing
  float maxDist = kernelSize * 1.4142;
  uint32_t imageColor;
  if (sigmaImage) {
    imageColor = inputImages[baseOffset + y * size.x + x];
  }
  if (!extrapolation) {
    if (sigmaImage > 0) {
      if (!Image::RGBA::a(imageColor)) {
        outputFlow[y * size.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
        return;
      }
    }
  }
  float sumWeight = 0;
  float2 sumContribution = make_float2(0, 0);

  if (maxDist == 0) maxDist = 1;
  for (int t = 0; t < frameCount; t++)
    if (frames[t] >= 0) {
      for (int dx = -kernelSize; dx <= kernelSize; dx++)
        for (int dy = -kernelSize; dy <= kernelSize; dy++) {
          const int offset = t * size.x * size.y;
          // Here i came across a neighbor, what he is look like
          int2 neighborCoord = make_int2(x + dx, y + dy);
          if (!inRange(neighborCoord, size)) {
            continue;
          }
          float neighborConfidence = 1;
          if (inputConfidences) {
            neighborConfidence = inputConfidences[offset + neighborCoord.y * size.x + neighborCoord.x];
          }
          if (neighborConfidence == 0) {
            continue;
          }
          float2 neighborflowOffset = inputFlows[offset + neighborCoord.y * size.x + neighborCoord.x];
          if (neighborflowOffset.x == INVALID_FLOW_VALUE) {
            continue;
          }
          float weightImage = 1.0f;
          if (sigmaImage) {
            uint32_t imageColorNeighbor = inputImages[offset + neighborCoord.y * size.x + neighborCoord.x];
            if (Image::RGBA::a(imageColorNeighbor) > 0 && Image::RGBA::a(imageColor) > 0) {
              const float sad = abs((float(Image::RGBA::r(imageColorNeighbor)) - Image::RGBA::r(imageColor)) / 255.0) +
                                abs((float(Image::RGBA::g(imageColorNeighbor)) - Image::RGBA::g(imageColor)) / 255.0) +
                                abs((float(Image::RGBA::b(imageColorNeighbor)) - Image::RGBA::b(imageColor)) / 255.0);
              weightImage = exp(-abs(sad * sad * sigmaImage));
            }
          }
          // Now calculate the distance of time
          float distTime = float(frames[t] - frames[frameId]) / frameCount;
          float weightTime = exp(-abs(distTime * distTime * sigmaTime));
          // Now calculate the distance between source and target
          float distSpace = length(make_float2(dx, dy)) / maxDist;
          float weightSpace = exp(-abs(distSpace * distSpace * sigmaSpace));
          // Now i do really look at the neighbor on the other side to see  how think is going on there
          float weight = weightSpace * weightImage * weightTime * neighborConfidence;
          sumWeight += weight;
          sumContribution += weight * neighborflowOffset;
        }
    }
  // If my confidence is high, i would tend to keep mine, don't care about the neighbor's confidence
  // Here is where to set the weight
  if (sumWeight == 0) {
    outputFlow[y * size.x + x] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
  } else {
    outputFlow[y * size.x + x] = sumContribution / sumWeight;
  }
}

Status SimpleFlow::performTemporalAwareFlowInterpolation(
    const bool extrapolation, const frameid_t frameId, const int2 size, const int kernelSize, const float sigmaSpace,
    const float sigmaImage, const float sigmaTime, const GPU::Buffer<const float> frames,
    const GPU::Buffer<const uint32_t> inputImages, const GPU::Buffer<const float2> inputFlows,
    const GPU::Buffer<const float> inputConfidences, GPU::Buffer<float2> outputFlow, GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);
  const int frameCount = (int)frames.numElements();
  temporalAwareFlowBlurKernel<<<dimGrid, dimBlock, 0, stream>>>(
      extrapolation, frameId, frameCount, size, kernelSize, sigmaSpace, sigmaImage, sigmaTime, frames.get(),
      inputImages.get(), inputFlows.get(), inputConfidences.get(), outputFlow.get());
  return CUDA_STATUS;
}

Status SimpleFlow::performFlowJittering(const int jitterSize, const int windowSize, const float flowMagnitudeWeight,
                                        const float gradientWeight, const int2 size0, const int2 offset0,
                                        const GPU::Buffer<const uint32_t> inputBuffer0,
                                        const GPU::Buffer<const float> inputGradientBuffer0, const int2 size1,
                                        const int2 offset1, const GPU::Buffer<const uint32_t> inputBuffer1,
                                        const GPU::Buffer<const float> inputGradientBuffer1,
                                        const GPU::Buffer<const float2> inputFlow, GPU::Buffer<float2> outputFlow,
                                        GPU::Stream gpuStream) {
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(SIMPLEFLOW_KERNEL_BLOCK_SIZE_X, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size0.x, SIMPLEFLOW_KERNEL_BLOCK_SIZE_X),
               (unsigned)Cuda::ceilDiv(size0.y, SIMPLEFLOW_KERNEL_BLOCK_SIZE_Y), 1);

  forwardFlowKernel<<<dimGrid, dimBlock, 0, stream>>>(jitterSize, windowSize, flowMagnitudeWeight, gradientWeight,
                                                      size0, offset0, inputBuffer0.get(), inputGradientBuffer0.get(),
                                                      size1, offset1, inputBuffer1.get(), inputGradientBuffer1.get(),
                                                      inputFlow.get(), outputFlow.get(), 0);
  return CUDA_STATUS;
}

}  // namespace Core
}  // namespace VideoStitch
