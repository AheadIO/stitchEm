#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "parallax/flowSequence.hpp"

#include "./kernels/patchDifferenceFunction.cu"

#include "backend/common/vectorOps.hpp"
#include "backend/cuda/deviceBuffer.hpp"
#include "backend/cuda/deviceStream.hpp"
#include "cuda/error.hpp"
#include "cuda/util.hpp"
#include "gpu/image/sampling.hpp"
#include "gpu/image/imageOps.hpp"
#include "gpu/image/blur.hpp"
#include "gpu/stream.hpp"

#include <string.h>

namespace VideoStitch {
namespace Core {
#define WARPER_BLOCK_SIZE_X 16
#define WARPER_BLOCK_SIZE_Y 16
#define WARPER_BLOCK_SIZE_Z 16

__global__ void weightedAvgFlowWarpKernel(const int2 size, const int frameId, const int frameCount,
                                          const float sigmaTime, const float* frames, const float2* inputFlows,
                                          float2* outputFlow) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < size.x && y < size.y) {
    const int index = y * size.x + x;
    float2 avgFlow = make_float2(0, 0);
    float totalWeight = 0.0;
    for (int t = 0; t < frameCount; t++)
      if (frames[t] >= 0) {
        // Now calculate the distance of time
        float distTime = float(frames[t] - frames[frameId]) / frameCount;
        float weightTime = exp(-abs(distTime * distTime * sigmaTime));
        const float2 inputFlow = inputFlows[t * size.x * size.y + index];
        if (inputFlow.x != INVALID_FLOW_VALUE) {
          avgFlow += inputFlow * weightTime;
          totalWeight += weightTime;
        }
      }
    if (totalWeight > 0) {
      outputFlow[index] = avgFlow / totalWeight;
    } else {
      outputFlow[index] = make_float2(INVALID_FLOW_VALUE, INVALID_FLOW_VALUE);
    }
  }
}

Status FlowSequence::regularizeFlowTemporally(const std::string& name, const frameid_t frame, const int2 size,
                                              const int2 offset, GPU::Buffer<float2> flow, GPU::Stream gpuStream) {
  // Cache the input flow
  FAIL_RETURN(cacheBuffer<float2>(frame, name, size, offset, flow, gpuStream));

  TypedCached<float2>* cache = dynamic_cast<TypedCached<float2>*>(getFlowCachedBuffer(name).get());
  if (!cache) {
    return {Origin::ImageFlow, ErrType::InvalidConfiguration, "FlowSequence::cache is not valid"};
  }
  const int frameIndex = getFrameIndex(frame);
  if (frameIndex < 0) {
    return {Origin::ImageFlow, ErrType::InvalidConfiguration, "FlowSequence::frameindex < 0"};
  }

  // Now compute the weighted average flow
  // Now make the flow as stable as possible from previous computation
  hipStream_t stream = gpuStream.get();
  dim3 dimBlock(WARPER_BLOCK_SIZE_X, WARPER_BLOCK_SIZE_Y, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(size.x, dimBlock.x), (unsigned)Cuda::ceilDiv(size.x, dimBlock.y), 1);

  weightedAvgFlowWarpKernel<<<dimGrid, dimBlock, 0, stream>>>(size, frameIndex, (int)getFrames().numElements(), 5,
                                                              getFrames().get(), cache->getBuffer().get(), flow.get());
  return CUDA_STATUS;
}
}  // namespace Core
}  // namespace VideoStitch
