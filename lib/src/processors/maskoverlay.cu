#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "gpu/processors/maskoverlay.hpp"

#include "backend/common/imageOps.hpp"

#include "backend/cuda/surface.hpp"
#include "backend/cuda/deviceStream.hpp"
#include "cuda/util.hpp"

namespace VideoStitch {
namespace Core {

namespace {
/**
 * A kernel that overlays the mask over the image.
 */
__global__ void maskOverlayKernel(hipSurfaceObject_t dst, unsigned width, unsigned height, int32_t r, int32_t g,
                                  int32_t b, int32_t alpha) {
  const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    uint32_t srcColor;
    surf2Dread(&srcColor, dst, x * sizeof(uint32_t), y);

    float overlayAlpha = float(alpha) / 255.f;

    if (VideoStitch::Image::RGBA::a(srcColor) < 255) {
      const int32_t overlayR = VideoStitch::Image::RGBA::r(srcColor) * (1 - overlayAlpha) + overlayAlpha * r;
      const int32_t overlayG = VideoStitch::Image::RGBA::g(srcColor) * (1 - overlayAlpha) + overlayAlpha * g;
      const int32_t overlayB = VideoStitch::Image::RGBA::b(srcColor) * (1 - overlayAlpha) + overlayAlpha * b;

      surf2Dwrite(VideoStitch::Image::RGBA::pack(overlayR, overlayG, overlayB, 255), dst, x * sizeof(uint32_t), y);
    }
  }
}
}  // namespace

Status maskOverlay(GPU::Surface& dst, unsigned width, unsigned height, uint32_t color, GPU::Stream stream) {
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y), 1);
  maskOverlayKernel<<<dimGrid, dimBlock, 0, stream.get()>>>(
      dst.get().surface(), width, height, VideoStitch::Image::RGBA::r(color), VideoStitch::Image::RGBA::g(color),
      VideoStitch::Image::RGBA::b(color), VideoStitch::Image::RGBA::a(color));
  return CUDA_STATUS;
}
}  // namespace Core
}  // namespace VideoStitch
