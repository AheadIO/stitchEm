#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#pragma once

#include "backend/common/core/transformPhotoParam.hpp"

#ifdef VS_OPENCL
#define HIP_PI_F 3.141592654f

#define __device__
#define __host__

#endif

#ifdef __CUDA_ARCH__
#define powf_photo_vs __powf
#define saturatef_vs __saturatef
#define float2int_rd_vs __float2int_rd
#else
#define powf_photo_vs powf
#define saturatef_vs(f) (f < 0.0f ? 0.0f : (f > 1.0f ? 1.0f : f))
#define float2int_rd_vs (int)
#endif

#include <cmath>

namespace VideoStitch {
namespace Core {

/**
 * Photometric response correction functions:
 */

struct LinearPhotoCorrection {
  // enum SHMem { SharedMemoryNeed = 0 };
  static inline __device__ __host__ const float* setup(const float* floatPtr) { return floatPtr; }
  static inline __device__ __host__ float3 corr(float3 c, float /* photoParam */, const float* /* floatPtr */) {
    return c;
  }
  static inline __device__ __host__ float3 invCorr(float3 c, float /* photoParam */, const float* /* floatPtr */) {
    return c;
  }
};

struct GammaPhotoCorrection {
  // enum SHMem { SharedMemoryNeed = 0 };
  static inline __device__ __host__ const float* setup(const float* floatPtr) { return floatPtr; }
  static inline __device__ __host__ float3 corr(float3 color, float gamma, const float* /* floatPtr */) {
    color.x = powf_photo_vs(color.x / 255.0f, gamma);
    color.y = powf_photo_vs(color.y / 255.0f, gamma);
    color.z = powf_photo_vs(color.z / 255.0f, gamma);
    return color;
  }

  static inline __device__ __host__ float3 invCorr(float3 color, float gamma, const float* /* floatPtr */) {
    const float invGamma = 1.0f / gamma;
    color.x = 255.0f * powf_photo_vs(color.x, invGamma);
    color.y = 255.0f * powf_photo_vs(color.y, invGamma);
    color.z = 255.0f * powf_photo_vs(color.z, invGamma);
    return color;
  }
};

struct EmorPhotoCorrection {
  // The parameter is a lookup table of size 1024 * 2 (direct then inverse)
  static inline __device__ __host__ const float* setup(const float* floatPtr) {
#ifdef __CUDA_ARCH__
    __shared__ float lut[2049];
    const unsigned threadId = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = threadId; i < 2049; i += blockDim.x * blockDim.y) {
      lut[i] = floatPtr[i];
    }
    __syncthreads();
    return lut;
#else
    return floatPtr;
#endif
  }

  /**
   * Lookup f (in [0;1] in a lookup table).
   */
  static inline __device__ __host__ float lookup(float f, const float* lookupTable) {
    // When f == 1.0, then we get:
    // f == 1023.0, i == 1023, x == 0.0, and i + 1 == 1024.
    // Therefore we must allocate 1025 floats and put something valid in lookupTable[1024]
    // (The value does not matter as long as it's not nan of inf, it's multiplied by 0.0.
    f = saturatef_vs(f) * 1023.0f;
    const int i = float2int_rd_vs(f);
    const float x = f - i;  // in [0;1]
    return (1.0f - x) * lookupTable[i] + x * lookupTable[i + 1];
  }

  static inline __device__ __host__ float3 corr(float3 color, float /* floatParam */, const float* lookupTable) {
    const float* floatPtr = lookupTable;
    floatPtr += 1024;
    color.x = lookup(color.x / 255.0f, floatPtr);
    color.y = lookup(color.y / 255.0f, floatPtr);
    color.z = lookup(color.z / 255.0f, floatPtr);
    return color;
  }

  static inline __device__ __host__ float3 invCorr(float3 color, float /* floatParam */, const float* lookupTable) {
    const float* floatPtr = lookupTable;
    color.x = 255.0f * lookup(color.x, floatPtr);
    color.y = 255.0f * lookup(color.y, floatPtr);
    color.z = 255.0f * lookup(color.z, floatPtr);
    return color;
  }
};
}  // namespace Core
}  // namespace VideoStitch

#undef powf_photo_vs
#undef saturatef_vs
#undef float2int_rd_vs
