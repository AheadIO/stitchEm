// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#ifndef CUDAWITHINSTACK_H_
#define CUDAWITHINSTACK_H_

#ifndef VS_OPENCL
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#endif

#include <cmath>

#ifndef __CUDACC__
#include <algorithm>
#include <math.h>
#endif

namespace VideoStitch {
namespace Core {

/**
 * Input must be within both texture and crop rect
 */
inline __device__ bool isWithinRect(const float2 uv, float width, float height) {
  return 0.0f <= uv.x && uv.x < width && 0.0f <= uv.y && uv.y < height;
}

inline __device__ bool isWithinCropRect(const float2 uv, float width, float height, float cLeft, float cRight,
                                        float cTop, float cBottom) {
  return 0.0f <= uv.x && uv.x < width && 0.0f <= uv.y && uv.y < height && cLeft <= uv.x && uv.x <= cRight &&
         cTop <= uv.y && uv.y <= cBottom;
}

inline __device__ bool isWithinCropCircle(const float2 uv, float width, float height, float cLeft, float cRight,
                                          float cTop, float cBottom) {
  const float centerX = (cRight + cLeft) / 2.0f;
  const float centerY = (cBottom + cTop) / 2.0f;
  const float radius = fminf(cRight - cLeft, cBottom - cTop) / 2.0f;
  return 0.0f <= uv.x && uv.x < width && 0.0f <= uv.y && uv.y < height &&
         (uv.x - centerX) * (uv.x - centerX) + (uv.y - centerY) * (uv.y - centerY) < radius * radius;
}

namespace TransformStack {

inline __host__ bool isWithinCropRect(const float2 uv, float width, float height, float cLeft, float cRight, float cTop,
                                      float cBottom) {
  return 0.0f <= uv.x && uv.x < width && 0.0f <= uv.y && uv.y < height && cLeft <= uv.x && uv.x < cRight &&
         cTop <= uv.y && uv.y < cBottom;
}

inline __host__ bool isWithinCropCircle(const float2 uv, float width, float height, float cLeft, float cRight,
                                        float cTop, float cBottom) {
  const float centerX = (cRight + cLeft) / 2.0f;
  const float centerY = (cBottom + cTop) / 2.0f;
  const float radius = fminf(cRight - cLeft, cBottom - cTop) / 2.0f;
  return 0.0f <= uv.x && uv.x < width && 0.0f <= uv.y && uv.y < height &&
         (uv.x - centerX) * (uv.x - centerX) + (uv.y - centerY) * (uv.y - centerY) < radius * radius;
}

}  // namespace TransformStack

#ifdef __HIPCC__
/**
 * Output cropper with a rectangular shape.
 */
struct OutputRectCropper {
 public:
  static inline __device__ bool isPanoPointVisible(int x, int y, int panoWidth, int panoHeight) { return true; }
};

/**
 * Output cropper with a circular shape.
 */
struct OutputCircleCropper {
 public:
  static inline __device__ bool isPanoPointVisible(int x, int y, int panoWidth, int panoHeight) {
    // We want to be in a frame where values are at the pixel centers instead of pixels top-left.
    // That would mean using (x + 0.5, y + 0.5) instead of (x, y). Since we want integers, we multiply everything by 2
    // to yield (2 * x + 1, 2 * y + 1).
    x = 2 * x + 1;
    y = 2 * y + 1;

    // The radius is simply the smallest of the semi-axis. For an odd size, we always ignore the last pixel.
    const int centerX = panoWidth & (~1);
    const int centerY = panoHeight & (~1);
    int radiusSquared = min(centerX, centerY);
    radiusSquared *= radiusSquared;
    return (x - centerX) * (x - centerX) + (y - centerY) * (y - centerY) <= radiusSquared;
  }
};
#endif

}  // namespace Core
}  // namespace VideoStitch

#endif
