#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include <hip/hip_runtime.h>
#include "../cuda/error.hpp"
#include "../cuda/util.hpp"

#include <backend/cuda/deviceStream.hpp>

#include "transpose.hpp"
#include <stdint.h>

#define CUDABLOCKSIZE 16

namespace VideoStitch {
namespace Image {

/**
 * Transpose the general (not necessarily square) matrix @src
 * width @srcWidth columns and @srcHeight rows into dst.
 * @srcWidth and @srcHeight must be multiples of the tile size.
 */
template <unsigned CudaBlockSize, typename T>
__global__ void transposeKernel(T *dst, const T *src, unsigned srcWidth, unsigned srcHeight) {
  __shared__ T tile[CudaBlockSize][CudaBlockSize + 1];

  unsigned xIndex = blockIdx.x * CudaBlockSize + threadIdx.x;
  unsigned yIndex = blockIdx.y * CudaBlockSize + threadIdx.y;
  unsigned tileInStart = xIndex + yIndex * srcWidth;

  xIndex = blockIdx.y * CudaBlockSize + threadIdx.x;
  yIndex = blockIdx.x * CudaBlockSize + threadIdx.y;
  unsigned tileOutStart = xIndex + yIndex * srcHeight;

  for (int i = 0; i < CudaBlockSize; i += CudaBlockSize) {
    tile[threadIdx.y + i][threadIdx.x] = src[tileInStart + i * srcWidth];
  }

  __syncthreads();

  for (int i = 0; i < CudaBlockSize; i += CudaBlockSize) {
    dst[tileOutStart + i * srcHeight] = tile[threadIdx.x][threadIdx.y + i];
  }
}

/**
 * Transpose the general (not necessarily square) matrix @src
 * width @srcWidth columns and @srcHeight rows into dst.
 * @srcWidth and @srcHeight are arbitrary.
 */
template <unsigned CudaBlockSize, typename T>
__global__ void transposeGenericKernel(T *dst, const T *src, unsigned srcWidth, unsigned srcHeight) {
  __shared__ T tile[CudaBlockSize][CudaBlockSize + 1];

  const unsigned xSrcIndex = blockIdx.x * CudaBlockSize + threadIdx.x;
  const unsigned ySrcIndex = blockIdx.y * CudaBlockSize + threadIdx.y;
  const unsigned tileInStart = xSrcIndex + ySrcIndex * srcWidth;

  const unsigned xDstIndex = blockIdx.y * CudaBlockSize + threadIdx.x;
  const unsigned yDstIndex = blockIdx.x * CudaBlockSize + threadIdx.y;
  const unsigned tileOutStart = xDstIndex + yDstIndex * srcHeight;

  if (xSrcIndex < srcWidth && ySrcIndex < srcHeight) {
    tile[threadIdx.y][threadIdx.x] = src[tileInStart];
  }

  __syncthreads();
  if (xDstIndex < srcHeight && yDstIndex < srcWidth) {
    dst[tileOutStart] = tile[threadIdx.x][threadIdx.y];
  }
}

template <typename T>
Status transpose(T *dst, const T *src, int64_t w, int64_t h, GPU::Stream &stream) {
  dim3 dimBlock(CUDABLOCKSIZE, CUDABLOCKSIZE);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(w, CUDABLOCKSIZE), (unsigned)Cuda::ceilDiv(h, CUDABLOCKSIZE));
  transposeGenericKernel<CUDABLOCKSIZE, T><<<dimGrid, dimBlock, 0, stream.get()>>>(dst, src, (unsigned)w, (unsigned)h);
  return CUDA_STATUS;
}

// explicit template instanciations
template Status transpose(uint32_t *dst, const uint32_t *src, int64_t w, int64_t h, GPU::Stream &stream);
template Status transpose(unsigned char *dst, const unsigned char *src, int64_t w, int64_t h, GPU::Stream &stream);
template Status transpose(float *dst, const float *src, int64_t w, int64_t h, GPU::Stream &stream);
template Status transpose(float2 *dst, const float2 *src, int64_t w, int64_t h, GPU::Stream &stream);
}  // namespace Image
}  // namespace VideoStitch
