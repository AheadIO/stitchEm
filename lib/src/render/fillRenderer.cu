#include "hip/hip_runtime.h"
// Copyright (c) 2012-2017 VideoStitch SAS
// Copyright (c) 2018 stitchEm

#include "fillRenderer.hpp"

#include "../cuda/util.hpp"
#include "cuda/error.hpp"

namespace VideoStitch {
namespace Render {

namespace {
/**
 * Fill rectangle kernel.
 */
__global__ void fillRectKernel(uint32_t* dst, uint32_t value, unsigned left, unsigned top, unsigned right,
                               unsigned bottom, unsigned bufferWidth) {
  const unsigned x = left + blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned y = top + blockIdx.y * blockDim.y + threadIdx.y;
  if (x < right && y < bottom) {
    dst[y * bufferWidth + x] = value;
  }
}

/**
 * Clamps a value between 0 and a maximum value.
 * @param value input value
 * @param maxValue max value
 */
int64_t clampZeroMax(int64_t value, int64_t maxValue) {
  if (value < 0) {
    return 0;
  } else if (value > maxValue) {
    return maxValue;
  }
  return value;
}
}  // namespace

Status FillRenderer::draw(uint32_t* dst, int64_t dstWidth, int64_t dstHeight, int64_t left, int64_t top, int64_t right,
                          int64_t bottom, uint32_t color, uint32_t /*bgcolor*/, hipStream_t stream) const {
  left = clampZeroMax(left, dstWidth);
  right = clampZeroMax(right, dstWidth);
  top = clampZeroMax(top, dstHeight);
  bottom = clampZeroMax(bottom, dstHeight);
  int64_t width = right - left;
  int64_t height = bottom - top;
  if (width <= 0 || height <= 0) {
    return {Origin::GPU, ErrType::ImplementationError, "Negative size for rectangle filling"};
  }
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid((unsigned)Cuda::ceilDiv(width, dimBlock.x), (unsigned)Cuda::ceilDiv(height, dimBlock.y));
  fillRectKernel<<<dimGrid, dimBlock, 0, stream>>>(dst, color, (unsigned)left, (unsigned)top, (unsigned)right,
                                                   (unsigned)bottom, (unsigned)dstWidth);
  return CUDA_STATUS;
}

}  // namespace Render
}  // namespace VideoStitch
